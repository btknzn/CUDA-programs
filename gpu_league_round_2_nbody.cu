#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////
////This is the code implementation for GPU Premier League Round 2: n-body simulation
//////////////////////////////////////////////////////////////////////////
#include <iostream>
#include <fstream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;

//////////////////////////////////////////////////////////////////////////
////TODO 0: Please replace the following strings with your team name and author names
////Note: Please do not use space in the string, use "_" instead
//////////////////////////////////////////////////////////////////////////

namespace name
{
	std::string team="using_namespace_std;";
	std::string author_1="Jeff Liu";
};

//////////////////////////////////////////////////////////////////////////
////Here is a sample function implemented on CPU for n-body simulation.

__host__ void N_Body_Simulation_CPU_Poorman(double* pos_x,double* pos_y,double* pos_z,		////position array
											double* vel_x,double* vel_y,double* vel_z,		////velocity array
											double* acl_x,double* acl_y,double* acl_z,		////acceleration array
											const double* mass,								////mass array
											const int n,									////number of particles
											const double dt,								////timestep
											const double epsilon_squared)					////epsilon to avoid 0-denominator
{		
	////Step 1: set particle accelerations to be zero
	memset(acl_x,0x00,sizeof(double)*n);
	memset(acl_y,0x00,sizeof(double)*n);
	memset(acl_z,0x00,sizeof(double)*n);

	////Step 2: traverse all particle pairs and accumulate gravitational forces for each particle from pairwise interactions
	for(int i=0;i<n;i++){
		for(int j=0;j<n;j++){
			////skip calculating force for itself
			if(i==j) continue;

			////r_ij=x_j-x_i
			double rx=pos_x[j]-pos_x[i];
			double ry=pos_y[j]-pos_y[i];
			double rz=pos_z[j]-pos_z[i];

			////a_ij=m_j*r_ij/(r+epsilon)^3, 
			////noticing that we ignore the gravitational coefficient (assuming G=1)
			double dis_squared=rx*rx+ry*ry+rz*rz;
			double one_over_dis_cube=1.0/pow(sqrt(dis_squared+epsilon_squared),3);
			double ax=mass[j]*rx*one_over_dis_cube;
			double ay=mass[j]*ry*one_over_dis_cube;
			double az=mass[j]*rz*one_over_dis_cube;

			////accumulate the force to the particle
			acl_x[i]+=ax;
			acl_y[i]+=ay;
			acl_z[i]+=az;
		}
	}

	////Step 3: explicit time integration to update the velocity and position of each particle
	for(int i=0;i<n;i++){
		////v_{t+1}=v_{t}+a_{t}*dt
		vel_x[i]+=acl_x[i]*dt;
		vel_y[i]+=acl_y[i]*dt;
		vel_z[i]+=acl_z[i]*dt;

		////x_{t+1}=x_{t}+v_{t}*dt
		pos_x[i]+=vel_x[i]*dt;
		pos_y[i]+=vel_y[i]*dt;
		pos_z[i]+=vel_z[i]*dt;
	}
}


//////////////////////////////////////////////////////////////////////////


__global__ void N_Body_Simulation_GPU(const double* pos_x, const double* pos_y, const double* pos_z,		////position array
									const double* vel_x, const double* vel_y, const double* vel_z,		////velocity array
									double* pos_x_dst, double* pos_y_dst, double* pos_z_dst,		////destination
                                    double* vel_x_dst, double* vel_y_dst, double* vel_z_dst,	    ////destination
                                    // acceleraion array unnecessary since I'm calculating in regisers
                                    const double* mass,								////mass array
                                    const int n,									////number of particles
                                    const double dt,								////timestep
                                    const double epsilon_squared)					////epsilon to avoid 0-denominator
{	
    // prepare dynamic shared memory
    extern __shared__ double data[];
    double *sharedX = &data[0];
    double *sharedY = &data[blockDim.x];
    double *sharedZ = &data[2*blockDim.x];
    double *sharedM = &data[3*blockDim.x];
	
    // prepare registers
	int arr_idx = blockIdx.x*blockDim.x + threadIdx.x;

    int shift_arr_idx;
    int shift_thread_idx;

    double dx; // save calculation time by saving these in registers
    double dy;
    double dz;

    double dis_squared; // registers for calculations
    double otherM;
    double one_over_dis_cube;

    double ax = 0; // acceleration
    double ay = 0;
    double az = 0;

    double vx = vel_x[arr_idx]; // velocity
    double vy = vel_y[arr_idx];
    double vz = vel_z[arr_idx];
    
    double px = pos_x[arr_idx]; // position
    double py = pos_y[arr_idx];
    double pz = pos_z[arr_idx];

    // here we go, begin cycling through tiles
    for (int blockShift = 0; blockShift < gridDim.x; ++blockShift) {
        // re-index with respect to blockShift, use modulo to loop back around
        shift_arr_idx = ((blockIdx.x + blockShift) & (gridDim.x-1)) * blockDim.x  + threadIdx.x;
		// load memory into shared
		// if (blockShift == 0) {
		// 	sharedX[threadIdx.x] = px;
		// 	sharedY[threadIdx.x] = py;
		// 	sharedZ[threadIdx.x] = pz;
		// 	sharedM[threadIdx.x] = mass[shift_arr_idx];
		// } else {
			sharedX[threadIdx.x] = pos_x[shift_arr_idx];
			sharedY[threadIdx.x] = pos_y[shift_arr_idx];
			sharedZ[threadIdx.x] = pos_z[shift_arr_idx];
			sharedM[threadIdx.x] = mass[shift_arr_idx];
		// }

		__syncthreads();
	
		#pragma unroll 64
        for (int threadShift = 0; threadShift < blockDim.x; ++threadShift) {
            // don't calculate your own gravity
			if (blockShift == 0 && threadShift == 0) continue;
			// if (blockShift == 0 && threadShift == threadIdx.x) continue;
            // shifted shared memory index that wraps around
            shift_thread_idx = (threadIdx.x + threadShift) & (blockDim.x-1);
            
            // // distance
            dx = sharedX[shift_thread_idx] - px;
            dy = sharedY[shift_thread_idx] - py;
			dz = sharedZ[shift_thread_idx] - pz;
			// dx = sharedX[threadShift] - px;
            // dy = sharedY[threadShift] - py;
			// dz = sharedZ[threadShift] - pz;
			// dx = 1;
			// dy = 1;
			// dz = 1;
            
            // calculations
            dis_squared = dx*dx + dy*dy + dz*dz;
			one_over_dis_cube = 1.0 / pow(sqrt(dis_squared + epsilon_squared), 3);
			// dis_squared = 2;
			// one_over_dis_cube = 2;
            // increment acceleration for one round
			otherM = sharedM[shift_thread_idx];
			// otherM = 100;
			// otherM = sharedM[threadShift];
			ax += otherM*dx*one_over_dis_cube;
			ay += otherM*dy*one_over_dis_cube;
            az += otherM*dz*one_over_dis_cube;
		}
		__syncthreads();
    }
    // all net forces and net accelerations done, now integrate velocity
    vx += ax * dt; 
    vy += ay * dt;
    vz += az * dt;
    // integrate position
    px += vx * dt; 
    py += vy * dt;
    pz += vz * dt;

    // write velocity to global memory
    vel_x_dst[arr_idx] = vx;
    vel_y_dst[arr_idx] = vy;
    vel_z_dst[arr_idx] = vz;
    // pos
    pos_x_dst[arr_idx] = px;
    pos_y_dst[arr_idx] = py;
    pos_z_dst[arr_idx] = pz;
}




__global__ void N_Body_Simulation_GPU_Unrolled(const double* pos_x, const double* pos_y, const double* pos_z,		////position array
	const double* vel_x, const double* vel_y, const double* vel_z,		////velocity array
	double* pos_x_dst, double* pos_y_dst, double* pos_z_dst,		////destination
	double* vel_x_dst, double* vel_y_dst, double* vel_z_dst,	    ////destination
	// acceleraion array unnecessary since I'm calculating in regisers
	const double* mass,								////mass array
	const int n,									////number of particles
	const double dt,								////timestep
	const double epsilon_squared)					////epsilon to avoid 0-denominator
{	
	// prepare dynamic shared memory
	extern __shared__ double data[];
	double *sharedX = &data[0];
	double *sharedY = &data[blockDim.x*2];
	double *sharedZ = &data[2*blockDim.x*2];
	double *sharedM = &data[3*blockDim.x*2];

	// prepare registers
	int arr_idx = blockIdx.x*2*blockDim.x + 2*threadIdx.x;

	int shift_arr_idx;
	int shift_thread_idx;

	double dx0; // save calculation time by saving these in registers
	double dy0;
	double dz0;

	double dis_squared0; // registers for calculations
	double otherM0;
	double one_over_dis_cube0;

	double ax0 = 0; // acceleration
	double ay0 = 0;
	double az0 = 0;

	double vx0 = vel_x[arr_idx]; // velocity
	double vy0 = vel_y[arr_idx];
	double vz0 = vel_z[arr_idx];

	double px0 = pos_x[arr_idx]; // position
	double py0 = pos_y[arr_idx];
	double pz0 = pos_z[arr_idx];

	double dx1; // save calculation time by saving these in registers
	double dy1;
	double dz1;

	double dis_squared1; // registers for calculations
	double otherM1;
	double one_over_dis_cube1;

	double ax1 = 0; // acceleration
	double ay1 = 0;
	double az1 = 0;

	double vx1 = vel_x[arr_idx + 1]; // velocity
	double vy1 = vel_y[arr_idx + 1];
	double vz1 = vel_z[arr_idx + 1];

	double px1 = pos_x[arr_idx + 1]; // position
	double py1 = pos_y[arr_idx + 1];
	double pz1 = pos_z[arr_idx + 1];


	// here we go, begin cycling through tiles
	for (int blockShift = 0; blockShift < gridDim.x; ++blockShift) {
		// re-index with respect to blockShift, use modulo to loop back around
		shift_arr_idx = ((blockIdx.x + blockShift)&(gridDim.x-1)) *2*blockDim.x + 2*threadIdx.x;
		// load memory into shared
		sharedX[2*threadIdx.x] = pos_x[shift_arr_idx];
		sharedY[2*threadIdx.x] = pos_y[shift_arr_idx];
		sharedZ[2*threadIdx.x] = pos_z[shift_arr_idx];
		sharedM[2*threadIdx.x] = mass[shift_arr_idx];

		sharedX[2*threadIdx.x + 1] = pos_x[shift_arr_idx + 1];
		sharedY[2*threadIdx.x + 1] = pos_y[shift_arr_idx + 1];
		sharedZ[2*threadIdx.x + 1] = pos_z[shift_arr_idx + 1];
		sharedM[2*threadIdx.x + 1] = mass[shift_arr_idx + 1];

		__syncthreads();

		#pragma unroll 32
        for (int threadShift = 0; threadShift < blockDim.x*2; threadShift += 2) {
            // don't calculate your own gravity
            if (blockShift == 0 && threadShift == 0) continue;
            // shifted shared memory index that wraps around
            shift_thread_idx = (2*threadIdx.x + threadShift) & (2*blockDim.x-1);
    
            // distance
            dx0 = sharedX[shift_thread_idx] - px0;
            dy0 = sharedY[shift_thread_idx] - py0;
			dz0 = sharedZ[shift_thread_idx] - pz0;
			
			dx1 = sharedX[shift_thread_idx + 1] - px1;
            dy1 = sharedY[shift_thread_idx + 1] - py1;
            dz1 = sharedZ[shift_thread_idx + 1] - pz1;
            
            // calculations
            dis_squared0 = dx0*dx0 + dy0*dy0 + dz0*dz0;
            one_over_dis_cube0 = 1.0 / pow(sqrt(dis_squared0 + epsilon_squared), 3);
            // increment acceleration for one round
            otherM0 = sharedM[shift_thread_idx];
			ax0 += otherM0*dx0*one_over_dis_cube0;
			ay0 += otherM0*dy0*one_over_dis_cube0;
			az0 += otherM0*dz0*one_over_dis_cube0;

			// calculations
            dis_squared1 = dx1*dx1 + dy1*dy1 + dz1*dz1;
            one_over_dis_cube1 = 1.0 / pow(sqrt(dis_squared1 + epsilon_squared), 3);
            // increment acceleration for one round
            otherM1 = sharedM[shift_thread_idx + 1];
			ax1 += otherM1*dx1*one_over_dis_cube1;
			ay1 += otherM1*dy1*one_over_dis_cube1;
			az1 += otherM1*dz1*one_over_dis_cube1;
		}
		__syncthreads();
	}
	// all net forces and net accelerations done, now integrate velocity
	vx0 += ax0 * dt; 
	vy0 += ay0 * dt;
	vz0 += az0 * dt;
	// integrate position
	px0 += vx0 * dt; 
	py0 += vy0 * dt;
	pz0 += vz0 * dt;

	// all net forces and net accelerations done, now integrate velocity
	vx1 += ax1 * dt; 
	vy1 += ay1 * dt;
	vz1 += az1 * dt;
	// integrate position
	px1 += vx1 * dt; 
	py1 += vy1 * dt;
	pz1 += vz1 * dt;

	// write velocity to global memory
	vel_x_dst[arr_idx] = vx0;
	vel_y_dst[arr_idx] = vy0;
	vel_z_dst[arr_idx] = vz0;
	// pos
	pos_x_dst[arr_idx] = px0;
	pos_y_dst[arr_idx] = py0;
	pos_z_dst[arr_idx] = pz0;
	// write velocity to global memory
	vel_x_dst[arr_idx + 1] = vx1;
	vel_y_dst[arr_idx + 1] = vy1;
	vel_z_dst[arr_idx + 1] = vz1;
	// pos
	pos_x_dst[arr_idx + 1] = px1;
	pos_y_dst[arr_idx + 1] = py1;
	pos_z_dst[arr_idx + 1] = pz1;
}


////Your implementations end here
//////////////////////////////////////////////////////////////////////////


//////////////////////////////////////////////////////////////////////////
////Test function for n-body simulator
ofstream out;

//////////////////////////////////////////////////////////////////////////
////Please do not change the values below
const double dt=0.001;							////time step
const int time_step_num=10;						////number of time steps
const double epsilon=1e-2;						////epsilon added in the denominator to avoid 0-division when calculating the gravitational force
const double epsilon_squared=epsilon*epsilon;	////epsilon squared

////We use grid_size=4 to help you debug your code, change it to a bigger number (e.g., 16, 32, etc.) to test the performance of your GPU code
const unsigned int grid_size=16;					////assuming particles are initialized on a background grid
const unsigned int particle_n=pow(grid_size,3);	////assuming each grid cell has one particle at the beginning

__host__ void Test_N_Body_Simulation()
{
	////initialize position, velocity, acceleration, and mass
	
	double* pos_x=new double[particle_n];
	double* pos_y=new double[particle_n];
	double* pos_z=new double[particle_n];
	////initialize particle positions as the cell centers on a background grid
	double dx=1.0/(double)grid_size;
	for(unsigned int k=0;k<grid_size;k++){
		for(unsigned int j=0;j<grid_size;j++){
			for(unsigned int i=0;i<grid_size;i++){
				unsigned int index=k*grid_size*grid_size+j*grid_size+i;
				pos_x[index]=dx*(double)i;
				pos_y[index]=dx*(double)j;
                pos_z[index]=dx*(double)k;
			}
		}
	}

	double* vel_x=new double[particle_n];
	memset(vel_x,0x00,particle_n*sizeof(double));
	double* vel_y=new double[particle_n];
	memset(vel_y,0x00,particle_n*sizeof(double));
	double* vel_z=new double[particle_n];
	memset(vel_z,0x00,particle_n*sizeof(double));

	double* acl_x=new double[particle_n];
	memset(acl_x,0x00,particle_n*sizeof(double));
	double* acl_y=new double[particle_n];
	memset(acl_y,0x00,particle_n*sizeof(double));
	double* acl_z=new double[particle_n];
	memset(acl_z,0x00,particle_n*sizeof(double));

	double* mass=new double[particle_n];
	for(int i=0;i<particle_n;i++){
		mass[i]=100.0;
	}
	cout.precision(6);
	#ifdef CPU
	//////////////////////////////////////////////////////////////////////////
	// Default implementation: n-body simulation on CPU
	// Comment the CPU implementation out when you test large-scale examples
	auto cpu_start=chrono::system_clock::now();
	cout<<"Total number of particles: "<<particle_n<<endl;
    cout<<"Tracking the motion of particle "<<particle_n/2<<endl;
	for(int i=0;i<time_step_num;i++){
		N_Body_Simulation_CPU_Poorman(pos_x,pos_y,pos_z,vel_x,vel_y,vel_z,acl_x,acl_y,acl_z,mass,particle_n,dt,epsilon_squared);
		cout<<"pos on timestep "<<fixed<<i<<": "<<pos_x[particle_n/2]<<", "<<pos_y[particle_n/2]<<", "<<pos_z[particle_n/2]<<endl;
        // cout<<"vel on timestep "<<i<<": "<<vel_x[particle_n/2]<<", "<<vel_y[particle_n/2]<<", "<<vel_z[particle_n/2]<<endl;
        // cout<<"acc on timestep "<<i<<": "<<acl_x[particle_n/2]<<", "<<acl_y[particle_n/2]<<", "<<acl_z[particle_n/2]<<endl;
    }
    // cout<<"pos on timestep "<<fixed<<time_step_num-1<<": "<<pos_x[particle_n/2]<<", "<<pos_y[particle_n/2]<<", "<<pos_z[particle_n/2]<<endl;

	auto cpu_end=chrono::system_clock::now();
	chrono::duration<double> cpu_time=cpu_end-cpu_start;
	cout<<"CPU runtime: "<<cpu_time.count()*1000.<<" ms."<<endl;
	#endif // CPU
	//////////////////////////////////////////////////////////////////////////


	// resetting starting positions before the timer begins
	pos_x=new double[particle_n];
	pos_y=new double[particle_n];
	pos_z=new double[particle_n];
	// hipHostMalloc((void **)&pos_x, particle_n*sizeof(double));
	// hipHostMalloc((void **)&pos_y, particle_n*sizeof(double));
	// hipHostMalloc((void **)&pos_z, particle_n*sizeof(double));
    for(unsigned int k=0;k<grid_size;k++){
		for(unsigned int j=0;j<grid_size;j++){
			for(unsigned int i=0;i<grid_size;i++){
				unsigned int index=k*grid_size*grid_size+j*grid_size+i;
				pos_x[index]=dx*(double)i;
				pos_y[index]=dx*(double)j;
				pos_z[index]=dx*(double)k;
				// cout << index << " " << pos_x[index] << endl;
			}
		}
    }

    // cout << pos_x[1] << " " << pos_y[1] << " " << pos_z[1] << endl;

	// hipHostMalloc((void **)&vel_x, particle_n*sizeof(double));
	// hipHostMalloc((void **)&vel_y, particle_n*sizeof(double));
	// hipHostMalloc((void **)&vel_z, particle_n*sizeof(double));
	vel_x=new double[particle_n];
	memset(vel_x,0x00,particle_n*sizeof(double));
	vel_y=new double[particle_n];
	memset(vel_y,0x00,particle_n*sizeof(double));
	vel_z=new double[particle_n];
	memset(vel_z,0x00,particle_n*sizeof(double));

	// // acl_x=new double[particle_n];
	// memset(acl_x,0x00,particle_n*sizeof(double));
	// // acl_y=new double[particle_n];
	// memset(acl_y,0x00,particle_n*sizeof(double));
	// // acl_z=new double[particle_n];
	// memset(acl_z,0x00,particle_n*sizeof(double));

	mass=new double[particle_n];
	// hipHostMalloc((void **)&mass, particle_n*sizeof(double));
	for(int i=0;i<particle_n;i++){
		mass[i]=100.0;
	}

	//////////////////////////////////////////////////////////////////////////
	////Your implementation: n-body simulator on GPU
	hipEvent_t start,end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	float gpu_time=0.0f;
	hipDeviceSynchronize();
	hipEventRecord(start);

	//////////////////////////////////////////////////////////////////////////
	////TODO 2: Your GPU functions are called here
	////Requirement: You need to copy data from the CPU arrays, conduct computations on the GPU, and copy the values back from GPU to CPU
	////The final positions should be stored in the same place as the CPU n-body function, i.e., pos_x, pos_y, pos_z
	////The correctness of your simulation will be evaluated by comparing the results (positions) with the results calculated by the default CPU implementations

	// double-buffering
    double *px_dev[2];
	hipMalloc((void **)&px_dev[0], particle_n*sizeof(double));
	hipMalloc((void **)&px_dev[1], particle_n*sizeof(double));
	hipMemcpyAsync(px_dev[0], pos_x, particle_n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpyAsync(px_dev[1], pos_x, particle_n*sizeof(double), hipMemcpyHostToDevice);
    double *py_dev[2];
	hipMalloc((void **)&py_dev[0], particle_n*sizeof(double));
	hipMalloc((void **)&py_dev[1], particle_n*sizeof(double));
	hipMemcpyAsync(py_dev[0], pos_y, particle_n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpyAsync(py_dev[1], pos_y, particle_n*sizeof(double), hipMemcpyHostToDevice);
    double *pz_dev[2];
    hipMalloc((void **)&pz_dev[0], particle_n*sizeof(double));
	hipMalloc((void **)&pz_dev[1], particle_n*sizeof(double));
	hipMemcpyAsync(pz_dev[0], pos_z, particle_n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpyAsync(pz_dev[1], pos_z, particle_n*sizeof(double), hipMemcpyHostToDevice);

	double *vx_dev[2];
	hipMalloc((void **)&vx_dev[0], particle_n*sizeof(double));
	hipMalloc((void **)&vx_dev[1], particle_n*sizeof(double));
	hipMemcpyAsync(vx_dev[0], vel_x, particle_n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpyAsync(vx_dev[1], vel_x, particle_n*sizeof(double), hipMemcpyHostToDevice);
    double *vy_dev[2];
	hipMalloc((void **)&vy_dev[0], particle_n*sizeof(double));
	hipMalloc((void **)&vy_dev[1], particle_n*sizeof(double));
	hipMemcpyAsync(vy_dev[0], vel_y, particle_n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpyAsync(vy_dev[1], vel_y, particle_n*sizeof(double), hipMemcpyHostToDevice);
    double *vz_dev[2];
    hipMalloc((void **)&vz_dev[0], particle_n*sizeof(double));
	hipMalloc((void **)&vz_dev[1], particle_n*sizeof(double));
	hipMemcpyAsync(vz_dev[0], vel_z, particle_n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpyAsync(vz_dev[1], vel_z, particle_n*sizeof(double), hipMemcpyHostToDevice);

    double *mass_dev;
    hipMalloc((void **)&mass_dev, particle_n*sizeof(double));
	hipMemcpyAsync(mass_dev, mass, particle_n*sizeof(double), hipMemcpyHostToDevice);
	// hipDeviceSynchronize();
    
	#ifdef GPU
	int blocksize = min(max(64, particle_n/32), 512);
	int gridsize = particle_n / blocksize;
	// blocksize /= 2;
	
	int src, dst;

	
	for (int i = 0;i < time_step_num; i++) {
		src = i & 1;
		dst = (src + 1) & 1;
		N_Body_Simulation_GPU<<<gridsize, blocksize, blocksize*4*sizeof(double)>>>(
			px_dev[src],py_dev[src],pz_dev[src],vx_dev[src],vy_dev[src],vz_dev[src],
			px_dev[dst],py_dev[dst],pz_dev[dst],vx_dev[dst],vy_dev[dst],vz_dev[dst],
			mass_dev,particle_n,dt,epsilon_squared);
		// N_Body_Simulation_GPU_Unrolled<<<gridsize, blocksize, blocksize*8*sizeof(double)>>>(
		// 	px_dev[src],py_dev[src],pz_dev[src],vx_dev[src],vy_dev[src],vz_dev[src],
		// 	px_dev[dst],py_dev[dst],pz_dev[dst],vx_dev[dst],vy_dev[dst],vz_dev[dst],
		// // 	mass_dev,particle_n,dt,epsilon_squared);
		// hipMemcpy(pos_x, px_dev[dst], particle_n*sizeof(double), hipMemcpyDeviceToHost);
    	// hipMemcpy(pos_y, py_dev[dst], particle_n*sizeof(double), hipMemcpyDeviceToHost);
    	// hipMemcpy(pos_z, pz_dev[dst], particle_n*sizeof(double), hipMemcpyDeviceToHost);
        // cout<<"pos on timestep "<<fixed<<i<<": "<<pos_x[particle_n/2]<<", "<<pos_y[particle_n/2]<<", "<<pos_z[particle_n/2]<<endl;
    }
	#endif // GPU

	hipMemcpyAsync(pos_x, px_dev[dst], particle_n*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpyAsync(pos_y, py_dev[dst], particle_n*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpyAsync(pos_z, pz_dev[dst], particle_n*sizeof(double), hipMemcpyDeviceToHost);
	// hipDeviceSynchronize();
	//////////////////////////////////////////////////////////////////////////

	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&gpu_time,start,end);
	printf("\nGPU runtime: %.4f ms\n",gpu_time);
	hipEventDestroy(start);
	hipEventDestroy(end);

	//////////////////////////////////////////////////////////////////////////

	cout<<"R0: "<<pos_x[particle_n/2]<<" " <<pos_y[particle_n/2]<<" " <<pos_z[particle_n/2]<<endl;
	out<<"T1: "<<gpu_time<<endl;
}

int main()
{
	if(name::team=="Team_X"){
		printf("\nPlease specify your team name and team member names in name::team and name::author to start.\n");
		return 0;
	}

	std::string file_name=name::team+"_competition_2_nbody.dat";
	out.open(file_name.c_str());
	
	if(out.fail()){
		printf("\ncannot open file %s to record results\n",file_name.c_str());
		return 0;
	}

	Test_N_Body_Simulation();

	return 0;
}
