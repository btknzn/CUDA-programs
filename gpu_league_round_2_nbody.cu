#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////
////This is the code implementation for GPU Premier League Round 2: n-body simulation
//////////////////////////////////////////////////////////////////////////
#include <iostream>
#include <fstream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;

//////////////////////////////////////////////////////////////////////////
////TODO 0: Please replace the following strings with your team name and author names
////Note: Please do not use space in the string, use "_" instead
//////////////////////////////////////////////////////////////////////////

namespace name
{
	std::string team="using_namespace_std;";
	std::string author_1="Jeff Liu";
};

//////////////////////////////////////////////////////////////////////////
////Here is a sample function implemented on CPU for n-body simulation.

__host__ void N_Body_Simulation_CPU_Poorman(double* pos_x,double* pos_y,double* pos_z,		////position array
											double* vel_x,double* vel_y,double* vel_z,		////velocity array
											double* acl_x,double* acl_y,double* acl_z,		////acceleration array
											const double* mass,								////mass array
											const int n,									////number of particles
											const double dt,								////timestep
											const double epsilon_squared)					////epsilon to avoid 0-denominator
{		
	////Step 1: set particle accelerations to be zero
	memset(acl_x,0x00,sizeof(double)*n);
	memset(acl_y,0x00,sizeof(double)*n);
	memset(acl_z,0x00,sizeof(double)*n);

	////Step 2: traverse all particle pairs and accumulate gravitational forces for each particle from pairwise interactions
	for(int i=0;i<n;i++){
		for(int j=0;j<n;j++){
			////skip calculating force for itself
			if(i==j) continue;

			////r_ij=x_j-x_i
			double rx=pos_x[j]-pos_x[i];
			double ry=pos_y[j]-pos_y[i];
			double rz=pos_z[j]-pos_z[i];

			////a_ij=m_j*r_ij/(r+epsilon)^3, 
			////noticing that we ignore the gravitational coefficient (assuming G=1)
			double dis_squared=rx*rx+ry*ry+rz*rz;
			double one_over_dis_cube=1.0/pow(sqrt(dis_squared+epsilon_squared),3);
			double ax=mass[j]*rx*one_over_dis_cube;
			double ay=mass[j]*ry*one_over_dis_cube;
			double az=mass[j]*rz*one_over_dis_cube;

			////accumulate the force to the particle
			acl_x[i]+=ax;
			acl_y[i]+=ay;
			acl_z[i]+=az;
		}
	}

	////Step 3: explicit time integration to update the velocity and position of each particle
	for(int i=0;i<n;i++){
		////v_{t+1}=v_{t}+a_{t}*dt
		vel_x[i]+=acl_x[i]*dt;
		vel_y[i]+=acl_y[i]*dt;
		vel_z[i]+=acl_z[i]*dt;

		////x_{t+1}=x_{t}+v_{t}*dt
		pos_x[i]+=vel_x[i]*dt;
		pos_y[i]+=vel_y[i]*dt;
		pos_z[i]+=vel_z[i]*dt;
	}
}


//////////////////////////////////////////////////////////////////////////

__global__ void N_Body_Simulation_GPU(double* pos_x,double* pos_y,double* pos_z,		////position array
                                    double* vel_x,double* vel_y,double* vel_z,		////velocity array
                                    // acceleraion array unnecessary since I'm calculating in regisers
                                    const double* mass,								////mass array
                                    const int n,									////number of particles
                                    const double dt,								////timestep
                                    const double epsilon_squared)					////epsilon to avoid 0-denominator
{	
    // prepare dynamic shared memory
    extern __shared__ double data[];
    double *sharedX = &data[0];
    double *sharedY = &data[blockDim.x];
    double *sharedZ = &data[2*blockDim.x];
    double *sharedM = &data[3*blockDim.x];

    // prepare registers
    int arr_idx = blockIdx.x*blockDim.x + threadIdx.x;
    int shift_arr_idx;
    int shift_thread_idx;

    double dx; // save calculation time by saving these in registers
    double dy;
    double dz;

    double dis_squared; // registers for calculations
    double otherM;
    double one_over_dis_cube;

    double ax = 0; // acceleration
    double ay = 0;
    double az = 0;

    double vx = vel_x[arr_idx]; // velocity
    double vy = vel_y[arr_idx];
    double vz = vel_z[arr_idx];
    
    double px = pos_x[arr_idx]; // position
    double py = pos_y[arr_idx];
    double pz = pos_z[arr_idx];

    // if (blockIdx.x * blockDim.x + threadIdx.x == 40) {
    //     printf("vx: %f, vy: %f, vz: %f, px: %f, py: %f, pz: %f\n", vx, vy, vz, px, py, pz);
    // }
    // printf("px: %f, py: %f, pz: %f, arr_idx: %d\n", px, py, pz, arr_idx);
    __syncthreads();
    // here we go, begin cycling through tiles
    for (int blockShift = 0; blockShift < gridDim.x; ++blockShift) {
        // re-index with respect to blockShift, use modulo to loop back around
        shift_arr_idx = ((blockIdx.x + blockShift)%gridDim.x) * blockDim.x  + threadIdx.x;
        // if (blockIdx.x * blockDim.x + threadIdx.x == 38) printf("shift %d, %d\n", blockShift, shift_arr_idx);
        // load memory into both register and shared
        // if (blockIdx.x * blockDim.x + threadIdx.x == 56 && blockShift == 2) {
        //     printf("myx: %f, myy: %f, myz: %f\n", myX, myY, myZ);
        // }
        sharedX[threadIdx.x] = pos_x[shift_arr_idx];
        sharedY[threadIdx.x] = pos_y[shift_arr_idx];
        sharedZ[threadIdx.x] = pos_z[shift_arr_idx];
        sharedM[threadIdx.x] = mass[shift_arr_idx];
        __syncthreads();

        for (int threadShift = 0; threadShift < blockDim.x; ++threadShift) {
            // don't calculate your own gravity
            if (blockShift == 0 && threadShift == 0) continue;
            // shifted shared memory index that wraps around
            shift_thread_idx = (threadIdx.x + threadShift) % blockDim.x;
            
            // if (blockIdx.x * blockDim.x + threadIdx.x == 47) printf("blockshift %d threadshift %d visited %d, %d\n", blockShift, threadShift, (blockIdx.x + blockShift)%gridDim.x, shift_thread_idx);
            // distance
            dx = sharedX[shift_thread_idx] - px;
            dy = sharedY[shift_thread_idx] - py;
            dz = sharedZ[shift_thread_idx] - pz;
            // if (blockIdx.x * blockDim.x + threadIdx.x == 40 && blockShift == 3 && threadShift == 14) {
            //     printf("dx: %f, dy: %f, dz: %f\n", dx, dy, dz);
            //     printf("px: %f, py: %f, pz: %f\n", px, py, pz);
            //     printf("sx: %f, sy: %f, sz: %f\n", sharedX[shift_thread_idx], sharedY[shift_thread_idx], sharedZ[shift_thread_idx]);
            // }
            // calculations
            dis_squared = dx*dx + dy*dy + dz*dz;
            one_over_dis_cube = 1.0 / pow(sqrt(dis_squared + epsilon_squared), 3);
            // increment acceleration for one round
            otherM = sharedM[shift_thread_idx];
			ax += otherM*dx*one_over_dis_cube;
			ay += otherM*dy*one_over_dis_cube;
            az += otherM*dz*one_over_dis_cube;
            __syncthreads();
        }
        __syncthreads();
    }
    // all net forces and net accelerations done, now integrate velocity
    vx += ax * dt; 
    vy += ay * dt;
    vz += az * dt;
    // integrate position
    px += vx * dt; 
    py += vy * dt;
    pz += vz * dt;

    // write velocity to global memory
    vel_x[arr_idx] = vx;
    vel_y[arr_idx] = vy;
    vel_z[arr_idx] = vz;
    // pos
    pos_x[arr_idx] = px;
    pos_y[arr_idx] = py;
    pos_z[arr_idx] = pz;
}

////Your implementations end here
//////////////////////////////////////////////////////////////////////////


//////////////////////////////////////////////////////////////////////////
////Test function for n-body simulator
ofstream out;

//////////////////////////////////////////////////////////////////////////
////Please do not change the values below
const double dt=0.001;							////time step
const int time_step_num=100;						////number of time steps
const double epsilon=1e-2;						////epsilon added in the denominator to avoid 0-division when calculating the gravitational force
const double epsilon_squared=epsilon*epsilon;	////epsilon squared

////We use grid_size=4 to help you debug your code, change it to a bigger number (e.g., 16, 32, etc.) to test the performance of your GPU code
const unsigned int grid_size=8;					////assuming particles are initialized on a background grid
const unsigned int particle_n=pow(grid_size,3);	////assuming each grid cell has one particle at the beginning

__host__ void Test_N_Body_Simulation()
{
	////initialize position, velocity, acceleration, and mass
	
	double* pos_x=new double[particle_n];
	double* pos_y=new double[particle_n];
	double* pos_z=new double[particle_n];
	////initialize particle positions as the cell centers on a background grid
	double dx=1.0/(double)grid_size;
	for(unsigned int k=0;k<grid_size;k++){
		for(unsigned int j=0;j<grid_size;j++){
			for(unsigned int i=0;i<grid_size;i++){
				unsigned int index=k*grid_size*grid_size+j*grid_size+i;
				pos_x[index]=dx*(double)i;
				pos_y[index]=dx*(double)j;
                pos_z[index]=dx*(double)k;
			}
		}
	}

	double* vel_x=new double[particle_n];
	memset(vel_x,0x00,particle_n*sizeof(double));
	double* vel_y=new double[particle_n];
	memset(vel_y,0x00,particle_n*sizeof(double));
	double* vel_z=new double[particle_n];
	memset(vel_z,0x00,particle_n*sizeof(double));

	double* acl_x=new double[particle_n];
	memset(acl_x,0x00,particle_n*sizeof(double));
	double* acl_y=new double[particle_n];
	memset(acl_y,0x00,particle_n*sizeof(double));
	double* acl_z=new double[particle_n];
	memset(acl_z,0x00,particle_n*sizeof(double));

	double* mass=new double[particle_n];
	for(int i=0;i<particle_n;i++){
		mass[i]=100.0;
	}


	//////////////////////////////////////////////////////////////////////////
	////Default implementation: n-body simulation on CPU
	////Comment the CPU implementation out when you test large-scale examples
	auto cpu_start=chrono::system_clock::now();
	cout<<"Total number of particles: "<<particle_n<<endl;
	cout<<"Tracking the motion of particle "<<particle_n/2<<endl;
	for(int i=0;i<time_step_num;i++){
		N_Body_Simulation_CPU_Poorman(pos_x,pos_y,pos_z,vel_x,vel_y,vel_z,acl_x,acl_y,acl_z,mass,particle_n,dt,epsilon_squared);
		cout<<"pos on timestep "<<i<<": "<<pos_x[particle_n/2]<<", "<<pos_y[particle_n/2]<<", "<<pos_z[particle_n/2]<<endl;
	}
	auto cpu_end=chrono::system_clock::now();
	chrono::duration<double> cpu_time=cpu_end-cpu_start;
	cout<<"CPU runtime: "<<cpu_time.count()*1000.<<" ms."<<endl;

	//////////////////////////////////////////////////////////////////////////


    // resetting starting positions before the timer begins
    for(unsigned int k=0;k<grid_size;k++){
		for(unsigned int j=0;j<grid_size;j++){
			for(unsigned int i=0;i<grid_size;i++){
				unsigned int index=k*grid_size*grid_size+j*grid_size+i;
				pos_x[index]=dx*(double)i;
				pos_y[index]=dx*(double)j;
				pos_z[index]=dx*(double)k;
			}
		}
    }

    // cout << pos_x[1] << " " << pos_y[1] << " " << pos_z[1] << endl;

	memset(vel_x,0x00,particle_n*sizeof(double));
	memset(vel_y,0x00,particle_n*sizeof(double));
	memset(vel_z,0x00,particle_n*sizeof(double));

	memset(acl_x,0x00,particle_n*sizeof(double));
	memset(acl_y,0x00,particle_n*sizeof(double));
	memset(acl_z,0x00,particle_n*sizeof(double));

	//////////////////////////////////////////////////////////////////////////
	////Your implementation: n-body simulator on GPU
	hipEvent_t start,end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	float gpu_time=0.0f;
	hipDeviceSynchronize();
	hipEventRecord(start);

	//////////////////////////////////////////////////////////////////////////
	////TODO 2: Your GPU functions are called here
	////Requirement: You need to copy data from the CPU arrays, conduct computations on the GPU, and copy the values back from GPU to CPU
	////The final positions should be stored in the same place as the CPU n-body function, i.e., pos_x, pos_y, pos_z
	////The correctness of your simulation will be evaluated by comparing the results (positions) with the results calculated by the default CPU implementations

    double *px_dev;
    hipMalloc((void **)&px_dev, particle_n*sizeof(double));
    hipMemcpy(px_dev, pos_x, particle_n*sizeof(double), hipMemcpyHostToDevice);
    
    double *py_dev;
    hipMalloc((void **)&py_dev, particle_n*sizeof(double));
    hipMemcpy(py_dev, pos_y, particle_n*sizeof(double), hipMemcpyHostToDevice);
    double *pz_dev;
    hipMalloc((void **)&pz_dev, particle_n*sizeof(double));
    hipMemcpy(pz_dev, pos_z, particle_n*sizeof(double), hipMemcpyHostToDevice);
    
    
    double *vx_dev;
    hipMalloc((void **)&vx_dev, particle_n*sizeof(double));
    hipMemcpy(vx_dev, vel_x, particle_n*sizeof(double), hipMemcpyHostToDevice);
    double *vy_dev;
    hipMalloc((void **)&vy_dev, particle_n*sizeof(double));
    hipMemcpy(vy_dev, vel_y, particle_n*sizeof(double), hipMemcpyHostToDevice);
    double *vz_dev;
    hipMalloc((void **)&vz_dev, particle_n*sizeof(double));
    hipMemcpy(vz_dev, vel_z, particle_n*sizeof(double), hipMemcpyHostToDevice);
    double *mass_dev;
    hipMalloc((void **)&mass_dev, particle_n*sizeof(double));
    hipMemcpy(mass_dev, mass, particle_n*sizeof(double), hipMemcpyHostToDevice);

    

    int blocksize = max(32, particle_n/32);
    int gridsize = particle_n / blocksize;
    // cout << blocksize << endl;
    // cout << gridsize << endl;

	for(int i=0;i<time_step_num;i++){
        N_Body_Simulation_GPU<<<gridsize, blocksize, blocksize*4*sizeof(double)>>>(px_dev,py_dev,pz_dev,vx_dev,vy_dev,vz_dev,mass_dev,particle_n,dt,epsilon_squared);
        // cout<<"pos on timestep "<<i<<": "<<pos_x[particle_n/2]<<", "<<pos_y[particle_n/2]<<", "<<pos_z[particle_n/2]<<endl;
    }

    
    hipMemcpy(pos_x, px_dev, particle_n*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(pos_y, py_dev, particle_n*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(pos_z, pz_dev, particle_n*sizeof(double), hipMemcpyDeviceToHost);
    


	//////////////////////////////////////////////////////////////////////////

	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&gpu_time,start,end);
	printf("\nGPU runtime: %.4f ms\n",gpu_time);
	hipEventDestroy(start);
	hipEventDestroy(end);
	//////////////////////////////////////////////////////////////////////////

	out<<"R0: "<<pos_x[particle_n/2]<<" " <<pos_y[particle_n/2]<<" " <<pos_z[particle_n/2]<<endl;
	out<<"T1: "<<gpu_time<<endl;
}

int main()
{
	if(name::team=="Team_X"){
		printf("\nPlease specify your team name and team member names in name::team and name::author to start.\n");
		return 0;
	}

	std::string file_name=name::team+"_competition_2_nbody.dat";
	out.open(file_name.c_str());
	
	if(out.fail()){
		printf("\ncannot open file %s to record results\n",file_name.c_str());
		return 0;
	}

	Test_N_Body_Simulation();

	return 0;
}

