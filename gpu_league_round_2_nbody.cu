#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////
////This is the code implementation for GPU Premier League Round 2: n-body simulation
//////////////////////////////////////////////////////////////////////////
#include <iostream>
#include <fstream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;

//////////////////////////////////////////////////////////////////////////
////TODO 0: Please replace the following strings with your team name and author names
////Note: Please do not use space in the string, use "_" instead
//////////////////////////////////////////////////////////////////////////

namespace name
{
	std::string team="using_namespace_std;";
	std::string author_1="Jeff Liu";
};

//////////////////////////////////////////////////////////////////////////
////Here is a sample function implemented on CPU for n-body simulation.

__host__ void N_Body_Simulation_CPU_Poorman(double* pos_x,double* pos_y,double* pos_z,		////position array
											double* vel_x,double* vel_y,double* vel_z,		////velocity array
											double* acl_x,double* acl_y,double* acl_z,		////acceleration array
											const double* mass,								////mass array
											const int n,									////number of particles
											const double dt,								////timestep
											const double epsilon_squared)					////epsilon to avoid 0-denominator
{		
	////Step 1: set particle accelerations to be zero
	memset(acl_x,0x00,sizeof(double)*n);
	memset(acl_y,0x00,sizeof(double)*n);
	memset(acl_z,0x00,sizeof(double)*n);

	////Step 2: traverse all particle pairs and accumulate gravitational forces for each particle from pairwise interactions
	for(int i=0;i<n;i++){
		for(int j=0;j<n;j++){
			////skip calculating force for itself
			if(i==j) continue;

			////r_ij=x_j-x_i
			double rx=pos_x[j]-pos_x[i];
			double ry=pos_y[j]-pos_y[i];
			double rz=pos_z[j]-pos_z[i];

			////a_ij=m_j*r_ij/(r+epsilon)^3, 
			////noticing that we ignore the gravitational coefficient (assuming G=1)
			double dis_squared=rx*rx+ry*ry+rz*rz;
			double one_over_dis_cube=1.0/pow(sqrt(dis_squared+epsilon_squared),3);
			double ax=mass[j]*rx*one_over_dis_cube;
			double ay=mass[j]*ry*one_over_dis_cube;
			double az=mass[j]*rz*one_over_dis_cube;

			////accumulate the force to the particle
			acl_x[i]+=ax;
			acl_y[i]+=ay;
			acl_z[i]+=az;
		}
	}

	////Step 3: explicit time integration to update the velocity and position of each particle
	for(int i=0;i<n;i++){
		////v_{t+1}=v_{t}+a_{t}*dt
		vel_x[i]+=acl_x[i]*dt;
		vel_y[i]+=acl_y[i]*dt;
		vel_z[i]+=acl_z[i]*dt;

		////x_{t+1}=x_{t}+v_{t}*dt
		pos_x[i]+=vel_x[i]*dt;
		pos_y[i]+=vel_y[i]*dt;
		pos_z[i]+=vel_z[i]*dt;
	}
}


//////////////////////////////////////////////////////////////////////////


__global__ void N_Body_Simulation_GPU(const double* pos_x, const double* pos_y, const double* pos_z,		////position array
									const double* vel_x, const double* vel_y, const double* vel_z,		////velocity array
									double* pos_x_dst, double* pos_y_dst, double* pos_z_dst,		////destination
                                    double* vel_x_dst, double* vel_y_dst, double* vel_z_dst,	    ////destination
                                    // acceleraion array unnecessary since I'm calculating in regisers
                                    const double* mass,								////mass array
                                    const int n,									////number of particles
                                    const double dt,								////timestep
                                    const double epsilon_squared)					////epsilon to avoid 0-denominator
{	
    // prepare dynamic shared memory
    extern __shared__ double data[];
    double *sharedX = &data[0];
    double *sharedY = &data[blockDim.x];
    double *sharedZ = &data[2*blockDim.x];
    double *sharedM = &data[3*blockDim.x];
	
    // prepare registers
	int arr_idx = blockIdx.x*blockDim.x + threadIdx.x;

    int shift_arr_idx;
    int shift_thread_idx;

    double dx; // save calculation time by saving these in registers
    double dy;
    double dz;

    double dis_squared; // registers for calculations
    double otherM;
    double one_over_dis_cube;

    double ax = 0; // acceleration
    double ay = 0;
    double az = 0;

    double vx = vel_x[arr_idx]; // velocity
    double vy = vel_y[arr_idx];
    double vz = vel_z[arr_idx];
    
    double px = pos_x[arr_idx]; // position
    double py = pos_y[arr_idx];
    double pz = pos_z[arr_idx];

    // here we go, begin cycling through tiles
    for (int blockShift = 0; blockShift < gridDim.x; ++blockShift) {
        // re-index with respect to blockShift, use modulo to loop back around
        shift_arr_idx = ((blockIdx.x + blockShift) & (gridDim.x-1)) * blockDim.x + threadIdx.x;
		// load memory into shared
		sharedX[threadIdx.x] = pos_x[shift_arr_idx];
		sharedY[threadIdx.x] = pos_y[shift_arr_idx];
		sharedZ[threadIdx.x] = pos_z[shift_arr_idx];
		sharedM[threadIdx.x] = mass[shift_arr_idx];

		__syncthreads();
	
		#pragma unroll 64
        for (int threadShift = 0; threadShift < blockDim.x; ++threadShift) {
            // don't calculate your own gravity
			if (blockShift == 0 && threadShift == 0) continue;

            // shifted shared memory index that wraps around
            shift_thread_idx = (threadIdx.x + threadShift) & (blockDim.x-1);
            
            // // distance
            dx = sharedX[shift_thread_idx] - px;
            dy = sharedY[shift_thread_idx] - py;
			dz = sharedZ[shift_thread_idx] - pz;
            
            // calculations
            dis_squared = dx*dx + dy*dy + dz*dz;
			one_over_dis_cube = 1.0 / pow(sqrt(dis_squared + epsilon_squared), 3);

            // increment acceleration for one round
			otherM = sharedM[shift_thread_idx];
			ax += otherM*dx*one_over_dis_cube;
			ay += otherM*dy*one_over_dis_cube;
            az += otherM*dz*one_over_dis_cube;
		}
		__syncthreads();
    }
    // all net forces and net accelerations done, now integrate velocity
    vx += ax * dt; 
    vy += ay * dt;
    vz += az * dt;
    // integrate position
    px += vx * dt; 
    py += vy * dt;
    pz += vz * dt;

    // write velocity to global memory
    vel_x_dst[arr_idx] = vx;
    vel_y_dst[arr_idx] = vy;
    vel_z_dst[arr_idx] = vz;
    // pos
    pos_x_dst[arr_idx] = px;
    pos_y_dst[arr_idx] = py;
    pos_z_dst[arr_idx] = pz;
}


////Your implementations end here
//////////////////////////////////////////////////////////////////////////


//////////////////////////////////////////////////////////////////////////
////Test function for n-body simulator
ofstream out;

//////////////////////////////////////////////////////////////////////////
////Please do not change the values below
const double dt=0.001;							////time step
const int time_step_num=10;						////number of time steps
const double epsilon=1e-2;						////epsilon added in the denominator to avoid 0-division when calculating the gravitational force
const double epsilon_squared=epsilon*epsilon;	////epsilon squared

////We use grid_size=4 to help you debug your code, change it to a bigger number (e.g., 16, 32, etc.) to test the performance of your GPU code
const unsigned int grid_size=16;					////assuming particles are initialized on a background grid
const unsigned int particle_n=pow(grid_size,3);	////assuming each grid cell has one particle at the beginning

__host__ void Test_N_Body_Simulation()
{
	////initialize position, velocity, acceleration, and mass
	
	double* pos_x=new double[particle_n];
	double* pos_y=new double[particle_n];
	double* pos_z=new double[particle_n];
	////initialize particle positions as the cell centers on a background grid
	double dx=1.0/(double)grid_size;
	for(unsigned int k=0;k<grid_size;k++){
		for(unsigned int j=0;j<grid_size;j++){
			for(unsigned int i=0;i<grid_size;i++){
				unsigned int index=k*grid_size*grid_size+j*grid_size+i;
				pos_x[index]=dx*(double)i;
				pos_y[index]=dx*(double)j;
                pos_z[index]=dx*(double)k;
			}
		}
	}

	double* vel_x=new double[particle_n];
	memset(vel_x,0x00,particle_n*sizeof(double));
	double* vel_y=new double[particle_n];
	memset(vel_y,0x00,particle_n*sizeof(double));
	double* vel_z=new double[particle_n];
	memset(vel_z,0x00,particle_n*sizeof(double));

	double* acl_x=new double[particle_n];
	memset(acl_x,0x00,particle_n*sizeof(double));
	double* acl_y=new double[particle_n];
	memset(acl_y,0x00,particle_n*sizeof(double));
	double* acl_z=new double[particle_n];
	memset(acl_z,0x00,particle_n*sizeof(double));

	double* mass=new double[particle_n];
	for(int i=0;i<particle_n;i++){
		mass[i]=100.0;
	}
	
	#ifdef CPU
		//////////////////////////////////////////////////////////////////////////
		// Default implementation: n-body simulation on CPU
		// Comment the CPU implementation out when you test large-scale examples
		cout.precision(6);
		auto cpu_start=chrono::system_clock::now();
		cout<<"Total number of particles: "<<particle_n<<endl;
		cout<<"Tracking the motion of particle "<<particle_n/2<<endl;
		for(int i=0;i<time_step_num;i++){
			N_Body_Simulation_CPU_Poorman(pos_x,pos_y,pos_z,vel_x,vel_y,vel_z,acl_x,acl_y,acl_z,mass,particle_n,dt,epsilon_squared);
			cout<<"pos on timestep "<<fixed<<i<<": "<<pos_x[particle_n/2]<<", "<<pos_y[particle_n/2]<<", "<<pos_z[particle_n/2]<<endl;
		}

		auto cpu_end=chrono::system_clock::now();
		chrono::duration<double> cpu_time=cpu_end-cpu_start;
		cout<<"CPU runtime: "<<cpu_time.count()*1000.<<" ms."<<endl;
	
		////////////////////////////////////////////////////////////////////////
	#endif // CPU

	// resetting starting positions before the timer begins
	pos_x=new double[particle_n];
	pos_y=new double[particle_n];
	pos_z=new double[particle_n];
	// hipHostMalloc((void **)&pos_x, particle_n*sizeof(double));
	// hipHostMalloc((void **)&pos_y, particle_n*sizeof(double));
	// hipHostMalloc((void **)&pos_z, particle_n*sizeof(double));
    for(unsigned int k=0;k<grid_size;k++){
		for(unsigned int j=0;j<grid_size;j++){
			for(unsigned int i=0;i<grid_size;i++){
				unsigned int index=k*grid_size*grid_size+j*grid_size+i;
				pos_x[index]=dx*(double)i;
				pos_y[index]=dx*(double)j;
				pos_z[index]=dx*(double)k;
			}
		}
    }

	// hipHostMalloc((void **)&vel_x, particle_n*sizeof(double));
	// hipHostMalloc((void **)&vel_y, particle_n*sizeof(double));
	// hipHostMalloc((void **)&vel_z, particle_n*sizeof(double));
	vel_x=new double[particle_n];
	memset(vel_x,0x00,particle_n*sizeof(double));
	vel_y=new double[particle_n];
	memset(vel_y,0x00,particle_n*sizeof(double));
	vel_z=new double[particle_n];
	memset(vel_z,0x00,particle_n*sizeof(double));

	mass=new double[particle_n];
	// hipHostMalloc((void **)&mass, particle_n*sizeof(double));
	for(int i=0;i<particle_n;i++){
		mass[i]=100.0;
	}

	//////////////////////////////////////////////////////////////////////////
	////Your implementation: n-body simulator on GPU
	hipEvent_t start,end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	float gpu_time=0.0f;
	hipDeviceSynchronize();
	hipEventRecord(start);

	//////////////////////////////////////////////////////////////////////////
	////TODO 2: Your GPU functions are called here
	////Requirement: You need to copy data from the CPU arrays, conduct computations on the GPU, and copy the values back from GPU to CPU
	////The final positions should be stored in the same place as the CPU n-body function, i.e., pos_x, pos_y, pos_z
	////The correctness of your simulation will be evaluated by comparing the results (positions) with the results calculated by the default CPU implementations

	// double-buffering
    double *px_dev[2];
	hipMalloc((void **)&px_dev[0], particle_n*sizeof(double));
	hipMalloc((void **)&px_dev[1], particle_n*sizeof(double));
	hipMemcpyAsync(px_dev[0], pos_x, particle_n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpyAsync(px_dev[1], pos_x, particle_n*sizeof(double), hipMemcpyHostToDevice);
    double *py_dev[2];
	hipMalloc((void **)&py_dev[0], particle_n*sizeof(double));
	hipMalloc((void **)&py_dev[1], particle_n*sizeof(double));
	hipMemcpyAsync(py_dev[0], pos_y, particle_n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpyAsync(py_dev[1], pos_y, particle_n*sizeof(double), hipMemcpyHostToDevice);
    double *pz_dev[2];
    hipMalloc((void **)&pz_dev[0], particle_n*sizeof(double));
	hipMalloc((void **)&pz_dev[1], particle_n*sizeof(double));
	hipMemcpyAsync(pz_dev[0], pos_z, particle_n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpyAsync(pz_dev[1], pos_z, particle_n*sizeof(double), hipMemcpyHostToDevice);

	double *vx_dev[2];
	hipMalloc((void **)&vx_dev[0], particle_n*sizeof(double));
	hipMalloc((void **)&vx_dev[1], particle_n*sizeof(double));
	hipMemcpyAsync(vx_dev[0], vel_x, particle_n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpyAsync(vx_dev[1], vel_x, particle_n*sizeof(double), hipMemcpyHostToDevice);
    double *vy_dev[2];
	hipMalloc((void **)&vy_dev[0], particle_n*sizeof(double));
	hipMalloc((void **)&vy_dev[1], particle_n*sizeof(double));
	hipMemcpyAsync(vy_dev[0], vel_y, particle_n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpyAsync(vy_dev[1], vel_y, particle_n*sizeof(double), hipMemcpyHostToDevice);
    double *vz_dev[2];
    hipMalloc((void **)&vz_dev[0], particle_n*sizeof(double));
	hipMalloc((void **)&vz_dev[1], particle_n*sizeof(double));
	hipMemcpyAsync(vz_dev[0], vel_z, particle_n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpyAsync(vz_dev[1], vel_z, particle_n*sizeof(double), hipMemcpyHostToDevice);

    double *mass_dev;
    hipMalloc((void **)&mass_dev, particle_n*sizeof(double));
	hipMemcpyAsync(mass_dev, mass, particle_n*sizeof(double), hipMemcpyHostToDevice);
	// hipDeviceSynchronize();
	
	// calculate block size dynamically
	int blocksize = min(max(64, particle_n/32), 512);
	int gridsize = particle_n / blocksize;
	
	int src, dst; // buffer numbers
	for (int i = 0;i < time_step_num; i++) {
		src = i & 1;
		dst = (src + 1) & 1;
		N_Body_Simulation_GPU<<<gridsize, blocksize, blocksize*4*sizeof(double)>>>(
			px_dev[src],py_dev[src],pz_dev[src],vx_dev[src],vy_dev[src],vz_dev[src],
			px_dev[dst],py_dev[dst],pz_dev[dst],vx_dev[dst],vy_dev[dst],vz_dev[dst],
			mass_dev,particle_n,dt,epsilon_squared);
		// hipMemcpy(pos_x, px_dev[dst], particle_n*sizeof(double), hipMemcpyDeviceToHost);
    	// hipMemcpy(pos_y, py_dev[dst], particle_n*sizeof(double), hipMemcpyDeviceToHost);
    	// hipMemcpy(pos_z, pz_dev[dst], particle_n*sizeof(double), hipMemcpyDeviceToHost);
        // cout<<"pos on timestep "<<fixed<<i<<": "<<pos_x[particle_n/2]<<", "<<pos_y[particle_n/2]<<", "<<pos_z[particle_n/2]<<endl;
    }

	hipMemcpyAsync(pos_x, px_dev[dst], particle_n*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpyAsync(pos_y, py_dev[dst], particle_n*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpyAsync(pos_z, pz_dev[dst], particle_n*sizeof(double), hipMemcpyDeviceToHost);
	// hipDeviceSynchronize();
	//////////////////////////////////////////////////////////////////////////

	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&gpu_time,start,end);
	printf("\nGPU runtime: %.4f ms\n",gpu_time);
	hipEventDestroy(start);
	hipEventDestroy(end);

	//////////////////////////////////////////////////////////////////////////

	out<<"R0: "<<pos_x[particle_n/2]<<" " <<pos_y[particle_n/2]<<" " <<pos_z[particle_n/2]<<endl;
	out<<"T1: "<<gpu_time<<endl;
}

int main()
{
	if(name::team=="Team_X"){
		printf("\nPlease specify your team name and team member names in name::team and name::author to start.\n");
		return 0;
	}

	std::string file_name=name::team+"_competition_2_nbody.dat";
	out.open(file_name.c_str());
	
	if(out.fail()){
		printf("\ncannot open file %s to record results\n",file_name.c_str());
		return 0;
	}

	Test_N_Body_Simulation();

	return 0;
}
