#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////
////This is the code implementation for GPU Premier League Round 2: n-body simulation
//////////////////////////////////////////////////////////////////////////
#include <iostream>
#include <fstream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;

//////////////////////////////////////////////////////////////////////////
////TODO 0: Please replace the following strings with your team name and author names
////Note: Please do not use space in the string, use "_" instead
//////////////////////////////////////////////////////////////////////////

namespace name
{
	std::string team="using_namespace_std;";
	std::string author_1="Jeff Liu";
};

//////////////////////////////////////////////////////////////////////////
////Here is a sample function implemented on CPU for n-body simulation.

__host__ void N_Body_Simulation_CPU_Poorman(double* pos_x,double* pos_y,double* pos_z,		////position array
											double* vel_x,double* vel_y,double* vel_z,		////velocity array
											double* acl_x,double* acl_y,double* acl_z,		////acceleration array
											const double* mass,								////mass array
											const int n,									////number of particles
											const double dt,								////timestep
											const double epsilon_squared)					////epsilon to avoid 0-denominator
{		
	////Step 1: set particle accelerations to be zero
	memset(acl_x,0x00,sizeof(double)*n);
	memset(acl_y,0x00,sizeof(double)*n);
	memset(acl_z,0x00,sizeof(double)*n);

	////Step 2: traverse all particle pairs and accumulate gravitational forces for each particle from pairwise interactions
	for(int i=0;i<n;i++){
		for(int j=0;j<n;j++){
			////skip calculating force for itself
			if(i==j) continue;

			////r_ij=x_j-x_i
			double rx=pos_x[j]-pos_x[i];
			double ry=pos_y[j]-pos_y[i];
			double rz=pos_z[j]-pos_z[i];

			// if (i == 0) {
			// 	printf("idx: %d\n", j);
			// 	printf("dx: %f, dy: %f, dz: %f\n", rx, ry, rz);
            //     printf("px: %f, py: %f, pz: %f\n", pos_x[i], pos_y[i], pos_z[i]);
            //     printf("sx: %f, sy: %f, sz: %f\n", pos_x[j], pos_y[j], pos_z[j]);
			// 	printf("mass: %f\n", mass[j]);
			// }

			////a_ij=m_j*r_ij/(r+epsilon)^3, 
			////noticing that we ignore the gravitational coefficient (assuming G=1)
			double dis_squared=rx*rx+ry*ry+rz*rz;
			double one_over_dis_cube=1.0/pow(sqrt(dis_squared+epsilon_squared),3);
			double ax=mass[j]*rx*one_over_dis_cube;
			double ay=mass[j]*ry*one_over_dis_cube;
			double az=mass[j]*rz*one_over_dis_cube;

			////accumulate the force to the particle
			acl_x[i]+=ax;
			acl_y[i]+=ay;
			acl_z[i]+=az;
		}
	}

	////Step 3: explicit time integration to update the velocity and position of each particle
	for(int i=0;i<n;i++){
		////v_{t+1}=v_{t}+a_{t}*dt
		vel_x[i]+=acl_x[i]*dt;
		vel_y[i]+=acl_y[i]*dt;
		vel_z[i]+=acl_z[i]*dt;

		////x_{t+1}=x_{t}+v_{t}*dt
		pos_x[i]+=vel_x[i]*dt;
		pos_y[i]+=vel_y[i]*dt;
		pos_z[i]+=vel_z[i]*dt;
	}
}


//////////////////////////////////////////////////////////////////////////

__global__ void printer(double* pos_x, double* pos_y, double* pos_z, int length)
{
	for (int i = 0; i < length; i++) {
		printf("%d: %f, %f, %f\n", i, pos_x[i], pos_y[i], pos_z[i]);
	}
}



__global__ void N_Body_Simulation_GPU(const double* pos_x, const double* pos_y, const double* pos_z,		////position array
									const double* vel_x, const double* vel_y, const double* vel_z,		////velocity array
									double* pos_x_dst, double* pos_y_dst, double* pos_z_dst,		////destination
                                    double* vel_x_dst, double* vel_y_dst, double* vel_z_dst,	    ////destination
                                    // acceleraion array unnecessary since I'm calculating in regisers
                                    const double* mass,								////mass array
                                    const int n,									////number of particles
                                    const double dt,								////timestep
                                    const double epsilon_squared)					////epsilon to avoid 0-denominator
{	
    // prepare dynamic shared memory
    extern __shared__ double data[];
    double *sharedX = &data[0];
    double *sharedY = &data[blockDim.x];
    double *sharedZ = &data[2*blockDim.x];
    double *sharedM = &data[3*blockDim.x];
	
    // prepare registers
	int arr_idx = blockIdx.x*blockDim.x + threadIdx.x;
	// printf("%d\n", arr_idx);
    int shift_arr_idx;
    int shift_thread_idx;

    double dx; // save calculation time by saving these in registers
    double dy;
    double dz;

    double dis_squared; // registers for calculations
    double otherM;
    double one_over_dis_cube;

    double ax = 0; // acceleration
    double ay = 0;
    double az = 0;

    double vx = vel_x[arr_idx]; // velocity
    double vy = vel_y[arr_idx];
    double vz = vel_z[arr_idx];
    
    double px = pos_x[arr_idx]; // position
    double py = pos_y[arr_idx];
    double pz = pos_z[arr_idx];

    
    // here we go, begin cycling through tiles
    for (int blockShift = 0; blockShift < gridDim.x; ++blockShift) {
        // re-index with respect to blockShift, use modulo to loop back around
        shift_arr_idx = ((blockIdx.x + blockShift)%gridDim.x) * blockDim.x  + threadIdx.x;
		// load memory into shared
        sharedX[threadIdx.x] = pos_x[shift_arr_idx];
        sharedY[threadIdx.x] = pos_y[shift_arr_idx];
        sharedZ[threadIdx.x] = pos_z[shift_arr_idx];
		sharedM[threadIdx.x] = mass[shift_arr_idx];

		__syncthreads();
		// if (blockIdx.x * blockDim.x + threadIdx.x == 0) {
		// 	printf("index %d blockshift %d threadidx %d: %f, %f, %f\n", blockIdx.x * blockDim.x + threadIdx.x, blockShift, threadIdx.x, pos_x[shift_arr_idx], pos_y[shift_arr_idx], pos_z[shift_arr_idx]);
		// }
		
		#pragma unroll 16
        for (int threadShift = 0; threadShift < blockDim.x; ++threadShift) {
            // don't calculate your own gravity
            if (blockShift == 0 && threadShift == 0) continue;
            // shifted shared memory index that wraps around
            shift_thread_idx = (threadIdx.x + threadShift) % blockDim.x;
            
            // distance
            dx = sharedX[shift_thread_idx] - px;
            dy = sharedY[shift_thread_idx] - py;
            dz = sharedZ[shift_thread_idx] - pz;
            // if (blockIdx.x * blockDim.x + threadIdx.x == 0) {
			// 	printf("idx: %d\n", shift_arr_idx + shift_thread_idx);
            //     printf("dx: %f, dy: %f, dz: %f\n", dx, dy, dz);
            //     printf("px: %f, py: %f, pz: %f\n", px, py, pz);
			// 	printf("sx: %f, sy: %f, sz: %f\n", sharedX[shift_thread_idx], sharedY[shift_thread_idx], sharedZ[shift_thread_idx]);
			// 	printf("mass: %f\n", sharedM[shift_thread_idx]);
			// 	// printf("shift_arr_idx: %d, shift_thread_idx: %d\n", shift_arr_idx, shift_thread_idx);
				
            // }
            // __syncthreads();
            // calculations
            dis_squared = dx*dx + dy*dy + dz*dz;
            one_over_dis_cube = 1.0 / pow(sqrt(dis_squared + epsilon_squared), 3);
            // increment acceleration for one round
            otherM = sharedM[shift_thread_idx];
			ax += otherM*dx*one_over_dis_cube;
			ay += otherM*dy*one_over_dis_cube;
            az += otherM*dz*one_over_dis_cube;
            __syncthreads();
        }
    }
    // all net forces and net accelerations done, now integrate velocity
    vx += ax * dt; 
    vy += ay * dt;
    vz += az * dt;
    // integrate position
    px += vx * dt; 
    py += vy * dt;
    pz += vz * dt;

    // write velocity to global memory
    vel_x_dst[arr_idx] = vx;
    vel_y_dst[arr_idx] = vy;
    vel_z_dst[arr_idx] = vz;
    // pos
    pos_x_dst[arr_idx] = px;
    pos_y_dst[arr_idx] = py;
    pos_z_dst[arr_idx] = pz;

    // if (blockIdx.x * blockDim.x + threadIdx.x == n/2) {
    //     printf("vx: %f, vy: %f, vz: %f, px: %f, py: %f, pz: %f\n", vx, vy, vz, px, py, pz);
    //     printf("ax: %f, ay: %f, az: %f\n", ax, ay, az);
	// }
}


__global__ void N_Body_Simulation_GPU_Loop(volatile double* pos_x,volatile double* pos_y,volatile double* pos_z,		////position array
	volatile double* vel_x,volatile double* vel_y,volatile double* vel_z,		////velocity array
	// acceleraion array unnecessary since I'm calculating in regisers
	const volatile double* mass,								////mass array
	const int n,									////number of particles
	const double dt,								////timestep
	const double epsilon_squared,				////epsilon to avoid 0-denominator
	const int timesteps)					
{
	// prepare dynamic shared memory
	extern __shared__ double data[];
	volatile double *sharedX = &data[0];
	volatile double *sharedY = &data[blockDim.x];		
	volatile double *sharedZ = &data[2*blockDim.x];
	volatile double *sharedM = &data[3*blockDim.x];

	// prepare registers
	int arr_idx = blockIdx.x*blockDim.x + threadIdx.x;
	int shift_arr_idx;
	int shift_thread_idx;

	double dx; // save calculation time by saving these in registers
	double dy;
	double dz;

	double dis_squared; // registers for calculations
	double otherM;
	double one_over_dis_cube;

	double ax; // acceleration
	double ay;
	double az;

	double vx = vel_x[arr_idx]; // velocity
	double vy = vel_y[arr_idx];
	double vz = vel_z[arr_idx];

	double px = pos_x[arr_idx]; // position
	double py = pos_y[arr_idx];
	double pz = pos_z[arr_idx];

	for (int i = 0; i < timesteps; ++i) { // each timestep
		ax = 0;
		ay = 0;
		az = 0;
		__syncthreads();

		// here we go, begin cycling through tiles
		for (int blockShift = 0; blockShift < gridDim.x; ++blockShift) {
			// re-index with respect to blockShift, use modulo to loop back around
			shift_arr_idx = ((blockIdx.x + blockShift)%gridDim.x) * blockDim.x  + threadIdx.x;
			// load memory into shared
			sharedX[threadIdx.x] = pos_x[shift_arr_idx];
			sharedY[threadIdx.x] = pos_y[shift_arr_idx];
			sharedZ[threadIdx.x] = pos_z[shift_arr_idx];
			sharedM[threadIdx.x] = mass[shift_arr_idx];
			__syncthreads();

			#pragma unroll
			for (int threadShift = 0; threadShift < blockDim.x; ++threadShift) {
				// don't calculate your own gravity
				if (blockShift == 0 && threadShift == 0) continue;
				// shifted shared memory index that wraps around
				shift_thread_idx = (threadIdx.x + threadShift) % blockDim.x;

				// distance
				dx = sharedX[shift_thread_idx] - px;
				dy = sharedY[shift_thread_idx] - py;
				dz = sharedZ[shift_thread_idx] - pz;
				// if (blockIdx.x * blockDim.x + threadIdx.x == 0 && blockShift == 21 && threadShift == 67) {
				//     // printf("dx: %f, dy: %f, dz: %f\n", dx, dy, dz);
				//     // printf("px: %f, py: %f, pz: %f\n", px, py, pz);
				//     // printf("sx: %f, sy: %f, sz: %f\n", sharedX[shift_thread_idx], sharedY[shift_thread_idx], sharedZ[shift_thread_idx]);
				// 	printf("shift_arr_idx: %d, shift_thread_idx: %d\n", shift_arr_idx, shift_thread_idx);
				// }
				// __syncthreads();
				// calculations
				dis_squared = dx*dx + dy*dy + dz*dz;
				one_over_dis_cube = 1.0 / pow(sqrt(dis_squared + epsilon_squared), 3);
				// increment acceleration for one round
				otherM = sharedM[shift_thread_idx];
				ax += otherM*dx*one_over_dis_cube;
				ay += otherM*dy*one_over_dis_cube;
				az += otherM*dz*one_over_dis_cube;
				__syncthreads();
			}
		}
		// all net forces and net accelerations done, now integrate velocity
		vx += ax * dt; 
		vy += ay * dt;
		vz += az * dt;
		// integrate position
		px += vx * dt; 
		py += vy * dt;
		pz += vz * dt;

		// write velocity to global memory
		vel_x[arr_idx] = vx;
		vel_y[arr_idx] = vy;
		vel_z[arr_idx] = vz;
		// pos
		pos_x[arr_idx] = px;
		pos_y[arr_idx] = py;
		pos_z[arr_idx] = pz;
		// if (blockIdx.x * blockDim.x + threadIdx.x == n/2) {
		//     printf("vx: %f, vy: %f, vz: %f, px: %f, py: %f, pz: %f\n", vx, vy, vz, px, py, pz);
		//     printf("ax: %f, ay: %f, az: %f\n", ax, ay, az);
		// }
		__threadfence_system();
	}
}

////Your implementations end here
//////////////////////////////////////////////////////////////////////////


//////////////////////////////////////////////////////////////////////////
////Test function for n-body simulator
ofstream out;

//////////////////////////////////////////////////////////////////////////
////Please do not change the values below
const double dt=0.001;							////time step
const int time_step_num=10;						////number of time steps
const double epsilon=1e-2;						////epsilon added in the denominator to avoid 0-division when calculating the gravitational force
const double epsilon_squared=epsilon*epsilon;	////epsilon squared

////We use grid_size=4 to help you debug your code, change it to a bigger number (e.g., 16, 32, etc.) to test the performance of your GPU code
const unsigned int grid_size=16;					////assuming particles are initialized on a background grid
const unsigned int particle_n=pow(grid_size,3);	////assuming each grid cell has one particle at the beginning

__host__ void Test_N_Body_Simulation()
{
	////initialize position, velocity, acceleration, and mass
	
	double* pos_x=new double[particle_n];
	double* pos_y=new double[particle_n];
	double* pos_z=new double[particle_n];
	////initialize particle positions as the cell centers on a background grid
	double dx=1.0/(double)grid_size;
	for(unsigned int k=0;k<grid_size;k++){
		for(unsigned int j=0;j<grid_size;j++){
			for(unsigned int i=0;i<grid_size;i++){
				unsigned int index=k*grid_size*grid_size+j*grid_size+i;
				pos_x[index]=dx*(double)i;
				pos_y[index]=dx*(double)j;
                pos_z[index]=dx*(double)k;
			}
		}
	}

	double* vel_x=new double[particle_n];
	memset(vel_x,0x00,particle_n*sizeof(double));
	double* vel_y=new double[particle_n];
	memset(vel_y,0x00,particle_n*sizeof(double));
	double* vel_z=new double[particle_n];
	memset(vel_z,0x00,particle_n*sizeof(double));

	double* acl_x=new double[particle_n];
	memset(acl_x,0x00,particle_n*sizeof(double));
	double* acl_y=new double[particle_n];
	memset(acl_y,0x00,particle_n*sizeof(double));
	double* acl_z=new double[particle_n];
	memset(acl_z,0x00,particle_n*sizeof(double));

	double* mass=new double[particle_n];
	for(int i=0;i<particle_n;i++){
		mass[i]=100.0;
	}
	cout.precision(6);
	#ifdef CPU
	//////////////////////////////////////////////////////////////////////////
	// Default implementation: n-body simulation on CPU
	// Comment the CPU implementation out when you test large-scale examples
	auto cpu_start=chrono::system_clock::now();
	cout<<"Total number of particles: "<<particle_n<<endl;
    cout<<"Tracking the motion of particle "<<particle_n/2<<endl;
	for(int i=0;i<time_step_num;i++){
		N_Body_Simulation_CPU_Poorman(pos_x,pos_y,pos_z,vel_x,vel_y,vel_z,acl_x,acl_y,acl_z,mass,particle_n,dt,epsilon_squared);
		cout<<"pos on timestep "<<fixed<<i<<": "<<pos_x[particle_n/2]<<", "<<pos_y[particle_n/2]<<", "<<pos_z[particle_n/2]<<endl;
        // cout<<"vel on timestep "<<i<<": "<<vel_x[particle_n/2]<<", "<<vel_y[particle_n/2]<<", "<<vel_z[particle_n/2]<<endl;
        // cout<<"acc on timestep "<<i<<": "<<acl_x[particle_n/2]<<", "<<acl_y[particle_n/2]<<", "<<acl_z[particle_n/2]<<endl;
    }
    // cout<<"pos on timestep "<<fixed<<time_step_num-1<<": "<<pos_x[particle_n/2]<<", "<<pos_y[particle_n/2]<<", "<<pos_z[particle_n/2]<<endl;

	auto cpu_end=chrono::system_clock::now();
	chrono::duration<double> cpu_time=cpu_end-cpu_start;
	cout<<"CPU runtime: "<<cpu_time.count()*1000.<<" ms."<<endl;
	#endif // CPU
	//////////////////////////////////////////////////////////////////////////


	// resetting starting positions before the timer begins
	pos_x=new double[particle_n];
	pos_y=new double[particle_n];
	pos_z=new double[particle_n];
    for(unsigned int k=0;k<grid_size;k++){
		for(unsigned int j=0;j<grid_size;j++){
			for(unsigned int i=0;i<grid_size;i++){
				unsigned int index=k*grid_size*grid_size+j*grid_size+i;
				pos_x[index]=dx*(double)i;
				pos_y[index]=dx*(double)j;
				pos_z[index]=dx*(double)k;
				// cout << index << " " << pos_x[index] << endl;
			}
		}
    }

    // cout << pos_x[1] << " " << pos_y[1] << " " << pos_z[1] << endl;

	vel_x=new double[particle_n];
	memset(vel_x,0x00,particle_n*sizeof(double));
	vel_y=new double[particle_n];
	memset(vel_y,0x00,particle_n*sizeof(double));
	vel_z=new double[particle_n];
	memset(vel_z,0x00,particle_n*sizeof(double));

	acl_x=new double[particle_n];
	memset(acl_x,0x00,particle_n*sizeof(double));
	acl_y=new double[particle_n];
	memset(acl_y,0x00,particle_n*sizeof(double));
	acl_z=new double[particle_n];
	memset(acl_z,0x00,particle_n*sizeof(double));

	//////////////////////////////////////////////////////////////////////////
	////Your implementation: n-body simulator on GPU
	hipEvent_t start,end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	float gpu_time=0.0f;
	hipDeviceSynchronize();
	hipEventRecord(start);

	//////////////////////////////////////////////////////////////////////////
	////TODO 2: Your GPU functions are called here
	////Requirement: You need to copy data from the CPU arrays, conduct computations on the GPU, and copy the values back from GPU to CPU
	////The final positions should be stored in the same place as the CPU n-body function, i.e., pos_x, pos_y, pos_z
	////The correctness of your simulation will be evaluated by comparing the results (positions) with the results calculated by the default CPU implementations

	// double-buffering
    double *px_dev[2];
	hipMalloc((void **)&px_dev[0], particle_n*sizeof(double));
	hipMalloc((void **)&px_dev[1], particle_n*sizeof(double));
	hipMemcpy(px_dev[0], pos_x, particle_n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(px_dev[1], pos_x, particle_n*sizeof(double), hipMemcpyHostToDevice);
    double *py_dev[2];
	hipMalloc((void **)&py_dev[0], particle_n*sizeof(double));
	hipMalloc((void **)&py_dev[1], particle_n*sizeof(double));
	hipMemcpy(py_dev[0], pos_y, particle_n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(py_dev[1], pos_y, particle_n*sizeof(double), hipMemcpyHostToDevice);
    double *pz_dev[2];
    hipMalloc((void **)&pz_dev[0], particle_n*sizeof(double));
	hipMalloc((void **)&pz_dev[1], particle_n*sizeof(double));
	hipMemcpy(pz_dev[0], pos_z, particle_n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(pz_dev[1], pos_z, particle_n*sizeof(double), hipMemcpyHostToDevice);

	double *vx_dev[2];
	hipMalloc((void **)&vx_dev[0], particle_n*sizeof(double));
	hipMalloc((void **)&vx_dev[1], particle_n*sizeof(double));
	hipMemcpy(vx_dev[0], vel_x, particle_n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(vx_dev[1], vel_x, particle_n*sizeof(double), hipMemcpyHostToDevice);
    double *vy_dev[2];
	hipMalloc((void **)&vy_dev[0], particle_n*sizeof(double));
	hipMalloc((void **)&vy_dev[1], particle_n*sizeof(double));
	hipMemcpy(vy_dev[0], vel_y, particle_n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(vy_dev[1], vel_y, particle_n*sizeof(double), hipMemcpyHostToDevice);
    double *vz_dev[2];
    hipMalloc((void **)&vz_dev[0], particle_n*sizeof(double));
	hipMalloc((void **)&vz_dev[1], particle_n*sizeof(double));
	hipMemcpy(vz_dev[0], vel_z, particle_n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(vz_dev[1], vel_z, particle_n*sizeof(double), hipMemcpyHostToDevice);

    double *mass_dev;
    hipMalloc((void **)&mass_dev, particle_n*sizeof(double));
    hipMemcpy(mass_dev, mass, particle_n*sizeof(double), hipMemcpyHostToDevice);
    
	#ifdef GPU
	int blocksize = max(64, particle_n/32);
    int gridsize = particle_n / blocksize;
	
	int src, dst;

	for(int i=0;i<time_step_num;i++){
		src = i % 2;
		dst = (src + 1) % 2;
		N_Body_Simulation_GPU<<<gridsize, blocksize, blocksize*4*sizeof(double)>>>(
			px_dev[src],py_dev[src],pz_dev[src],vx_dev[src],vy_dev[src],vz_dev[src],
			px_dev[dst],py_dev[dst],pz_dev[dst],vx_dev[dst],vy_dev[dst],vz_dev[dst],
			mass_dev,particle_n,dt,epsilon_squared);
		// hipDeviceSynchronize();
		// hipMemcpy(pos_x, px_dev[dst], particle_n*sizeof(double), hipMemcpyDeviceToHost);
    	// hipMemcpy(pos_y, py_dev[dst], particle_n*sizeof(double), hipMemcpyDeviceToHost);
    	// hipMemcpy(pos_z, pz_dev[dst], particle_n*sizeof(double), hipMemcpyDeviceToHost);
        // cout<<"pos on timestep "<<fixed<<i<<": "<<pos_x[particle_n/2]<<", "<<pos_y[particle_n/2]<<", "<<pos_z[particle_n/2]<<endl;
    }
	#endif // GPU
	// N_Body_Simulation_GPU_Loop<<<gridsize, blocksize, blocksize*4*sizeof(double)>>>(px_dev,py_dev,pz_dev,vx_dev,vy_dev,vz_dev,mass_dev,particle_n,dt,epsilon_squared,time_step_num);

	hipMemcpy(pos_x, px_dev[dst], particle_n*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(pos_y, py_dev[dst], particle_n*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(pos_z, pz_dev[dst], particle_n*sizeof(double), hipMemcpyDeviceToHost);



	//////////////////////////////////////////////////////////////////////////

	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&gpu_time,start,end);
	printf("\nGPU runtime: %.4f ms\n",gpu_time);
	hipEventDestroy(start);
	hipEventDestroy(end);

	//////////////////////////////////////////////////////////////////////////

	cout<<"R0: "<<pos_x[particle_n/2]<<" " <<pos_y[particle_n/2]<<" " <<pos_z[particle_n/2]<<endl;
	out<<"T1: "<<gpu_time<<endl;
}

int main()
{
	if(name::team=="Team_X"){
		printf("\nPlease specify your team name and team member names in name::team and name::author to start.\n");
		return 0;
	}

	std::string file_name=name::team+"_competition_2_nbody.dat";
	out.open(file_name.c_str());
	
	if(out.fail()){
		printf("\ncannot open file %s to record results\n",file_name.c_str());
		return 0;
	}

	Test_N_Body_Simulation();

	return 0;
}
