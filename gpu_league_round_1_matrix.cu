//////////////////////////////////////////////////////////////////////////
////This is the code implementation for GPU Premier League Round 1
//////////////////////////////////////////////////////////////////////////
#include <iostream>
#include <fstream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;

//////////////////////////////////////////////////////////////////////////
////TODO 0: Please replace the following strings with your team name and author names
////Note: Please do not use space in the string, use "_" instead
//////////////////////////////////////////////////////////////////////////

namespace name
{
	std::string team="using_namespace_std;";
	std::string author_1="Jeff Liu";
};

////This is a matrix class to carry out linear algebra operations on both GPU and CPU
////It is the same as the sample code I showed in class on Week 3. 

////NOTICE: You do not have to change the implementation in this class. 
////But if you do want to change part of it for performance reasons, please let us known by writting a submission note on Canvas.

class Matrix{
public:
    int m=0;							////number of rows
    int n=0;							////number of columns
	vector<float> elements_on_host;		////we use a std::vector for the element array on host
    float* elements_on_dev=0;			////we use a pointer for the element array on device
	bool on_host=true;

	////constructors
	__host__ Matrix(){}

	__host__ Matrix(const int _m,const int _n,bool _on_host=true)
	{
		on_host=_on_host;
		if(on_host)Resize_On_Host(_m,_n);
		else Resize_On_Device(_m,_n);
	}

	////destructor
	__host__ ~Matrix()
	{
		if(!on_host&&elements_on_dev!=0) hipFree(elements_on_dev);		
	}

	////Resize on host or device
	__host__ void Resize_On_Host(const int _m,const int _n)
	{
		if(m==_m&&n==_n)return;
		m=_m;
		n=_n;
		elements_on_host.resize(m*n);
	}

	__host__ void Resize_On_Device(const int _m,const int _n)
	{
		if(m==_m&&n==_n)return;
		m=_m;
		n=_n;
		if(elements_on_dev!=0)hipFree(elements_on_dev);
		hipMalloc((void**)&elements_on_dev,m*n*sizeof(float));
	}

	////random access a matrix element
	inline __host__ float& operator() (const int i,const int j)
	{
		return elements_on_host[i*n+j];
	}

	inline __host__ const float& operator() (const int i,const int j) const
	{
		return elements_on_host[i*n+j];
	}

	////copy data with four cases (CPU->CPU, GPU->CPU, GPU->GPU, CPU->GPU)
	__host__ Matrix& operator= (const Matrix& mtx)
	{
		if(on_host&&mtx.on_host){
			Resize_On_Host(mtx.m,mtx.n);
			elements_on_host=mtx.elements_on_host;
		}
		else if(on_host&&!mtx.on_host){
			Resize_On_Host(mtx.m,mtx.n);
			hipMemcpy(&elements_on_host[0],mtx.elements_on_dev,m*n*sizeof(float),hipMemcpyDeviceToHost);
		}
		else if(!on_host&&!mtx.on_host){
			Resize_On_Device(mtx.m,mtx.n);
			hipMemcpy(elements_on_dev,mtx.elements_on_dev,mtx.m*n*sizeof(float),hipMemcpyDeviceToDevice);
		}
		else if(!on_host&&mtx.on_host){
			Resize_On_Device(mtx.m,mtx.n);
			hipMemcpy(elements_on_dev,&mtx.elements_on_host[0],m*n*sizeof(float),hipMemcpyHostToDevice);
		}
		return *this;
	}

	////print matrix elements on screen
	__host__ friend ostream & operator << (ostream &out,const Matrix &mtx)
	{
		if(!mtx.on_host)
			cout<<"Print for matrix on device is not supported."<<endl;

		for(int i=0;i<mtx.m;i++){
			for(int j=0;j<mtx.n;j++){
				out<<mtx(i,j)<<", ";
			}
			out<<std::endl;
		}
		return out;
	}
};

//////////////////////////////////////////////////////////////////////////
////Your tasks start!

////This is a sample implementation without using any memory hierarchy
////The function calculates C=A*B, with dimA=[Am,An], dimB=[Bm,Bn], dimC=[Am,bn], and An=Bm
__global__ void Matrix_Multiplication_AB_Kernel_Poorman(const float* Ae,const float* Be,float* Ce,const int Am,const int An,const int Bn)
{
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	int j=blockIdx.y*blockDim.y+threadIdx.y;

	float val=0.f;
	for(int k=0;k<An;k++)
		val+=Ae[i*An+k]*Be[k*Bn+j];
	Ce[i*Bn+j]=val;
} 

//////////////////////////////////////////////////////////////////////////
////Task 1: implement your fast matrix-matrix multiplication in the following kernel function.
////The function parameters are the same as the sample function:
////The function calculates C=A*B, with dimA=[Am,An], dimB=[Bm,Bn], dimC=[Am,bn], and An=Bm
//////////////////////////////////////////////////////////////////////////

__global__ void Matrix_Multiplication_AB_Kernel_Your_Version(const float* Ae,const float* Be,float* Ce,const int Am,const int An,const int Bn)
{
	// initialize memory
	const int block_size = 32;
	const int num_tiles = An / block_size;
	__shared__ float a_shared[block_size][block_size];
	__shared__ float b_shared[block_size][block_size];
	__shared__ float c_shared[block_size][block_size];

	// calculate 1d index of correct item on A, B, C
	int thr_per_block = blockDim.y * blockDim.x;
	int c_idx = blockIdx.y * gridDim.x * thr_per_block + threadIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
	c_shared[threadIdx.y][threadIdx.x] = 0; // set everything to zero just the first time
	int a_idx, b_idx;

	for (int tile = 0; tile < num_tiles; ++tile) {
		// want blockIdx.x to increment
		a_idx = blockIdx.y * num_tiles * thr_per_block + threadIdx.y * num_tiles * blockDim.x + tile * blockDim.x + threadIdx.x;
		// want blockIdx.y to increment
		b_idx = tile * gridDim.x * thr_per_block + threadIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
		a_shared[threadIdx.y][threadIdx.x] = Ae[a_idx];
		b_shared[threadIdx.y][threadIdx.x] = Be[b_idx];
		__syncthreads();

		// lmao loop unrolling time my dudes
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][0] * b_shared[0][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][1] * b_shared[1][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][2] * b_shared[2][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][3] * b_shared[3][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][4] * b_shared[4][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][5] * b_shared[5][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][6] * b_shared[6][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][7] * b_shared[7][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][8] * b_shared[8][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][9] * b_shared[9][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][10] * b_shared[10][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][11] * b_shared[11][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][12] * b_shared[12][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][13] * b_shared[13][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][14] * b_shared[14][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][15] * b_shared[15][threadIdx.x];

		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][16] * b_shared[16][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][17] * b_shared[17][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][18] * b_shared[18][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][19] * b_shared[19][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][20] * b_shared[20][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][21] * b_shared[21][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][22] * b_shared[22][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][23] * b_shared[23][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][24] * b_shared[24][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][25] * b_shared[25][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][26] * b_shared[26][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][27] * b_shared[27][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][28] * b_shared[28][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][29] * b_shared[29][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][30] * b_shared[30][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += a_shared[threadIdx.y][31] * b_shared[31][threadIdx.x];
		__syncthreads();
	}
	// save to global
	Ce[c_idx] = c_shared[threadIdx.y][threadIdx.x];
}

////This is a sample implementation without using any memory hierarchy
////The function calculates the matrix multiplication, with C=A^T*B*A, A^T is the transpose of A, dimA=[Am,An], dimB=[Am,Am], and dimC=[An,An]
__global__ void Matrix_Multiplication_ATBA_Kernel_Poorman(const float* Ae,const float* Be,float* Ce,const int Am,const int An)
{
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	int j=blockIdx.y*blockDim.y+threadIdx.y;
	
	float val=0.f;
	for(int l=0;l<Am;l++)
		for(int k=0;k<Am;k++)
			val+=Ae[l*An+i]*Be[l*Am+k]*Ae[k*An+j];
	Ce[i*An+j]=val;
}

//////////////////////////////////////////////////////////////////////////
////Task 2: calculate the matrix multiplication in the following kernel function. 
////The function parameters are the same as the sample function:
////The function calculates the matrix multiplication, with C=A^T*B*A, A^T is the transpose of A, dimA=[Am,An], dimB=[Am,Am], and dimC=[An,An]
//////////////////////////////////////////////////////////////////////////

__global__ void Matrix_Multiplication_ATBA_Kernel_Your_Version(const float* Ae,const float* Be,float* Ce,const int Am,const int An)
{
	// memory setup
	const int num_tiles = Am / 32;
	__shared__ float aTT_shared[32][32];
	__shared__ float b_shared[32][32];
	__shared__ float a_shared[32][32];
	__shared__ float accum_shared[32][32];
	__shared__ float c_shared[32][32];

	// coordinate setup
	int thr_per_block = blockDim.y * blockDim.x;
	int c_idx = blockIdx.y*gridDim.x*thr_per_block + threadIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
	int a_idx, b_idx, aTT_idx;

	// initialize memory
	c_shared[threadIdx.y][threadIdx.x] = 0;

	// following psuedocode coordinates are (y,x)
	for (int ay = 0; ay < num_tiles; ++ay) { //ay = bx
		// load a(ay,blockIdx.x)
		a_idx = ay*gridDim.x*thr_per_block + threadIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
		a_shared[threadIdx.y][threadIdx.x] = Ae[a_idx];
		// clear accumulator
		accum_shared[threadIdx.y][threadIdx.x] = 0;
		__syncthreads();
		for (int by = 0; by < num_tiles; ++by) { // by = aTx = aTTy
			// calculate indices
			b_idx = by*num_tiles*thr_per_block + threadIdx.y*num_tiles*blockDim.x + ay*blockDim.x + threadIdx.x;
			aTT_idx = by*gridDim.x*thr_per_block + threadIdx.y*gridDim.x*blockDim.x + blockIdx.y*blockDim.x + threadIdx.x;

			// load aTT(by, blockIdx.y) (since we load A but column access) and b(by,ay)
			b_shared[threadIdx.y][threadIdx.x] = Be[b_idx];
			aTT_shared[threadIdx.y][threadIdx.x] = Ae[aTT_idx];
			__syncthreads();

			// multiply aT x b, accumulate
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[0][threadIdx.y] * b_shared[0][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[1][threadIdx.y] * b_shared[1][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[2][threadIdx.y] * b_shared[2][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[3][threadIdx.y] * b_shared[3][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[4][threadIdx.y] * b_shared[4][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[5][threadIdx.y] * b_shared[5][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[6][threadIdx.y] * b_shared[6][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[7][threadIdx.y] * b_shared[7][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[8][threadIdx.y] * b_shared[8][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[9][threadIdx.y] * b_shared[9][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[10][threadIdx.y] * b_shared[10][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[11][threadIdx.y] * b_shared[11][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[12][threadIdx.y] * b_shared[12][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[13][threadIdx.y] * b_shared[13][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[14][threadIdx.y] * b_shared[14][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[15][threadIdx.y] * b_shared[15][threadIdx.x];

			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[16][threadIdx.y] * b_shared[16][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[17][threadIdx.y] * b_shared[17][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[18][threadIdx.y] * b_shared[18][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[19][threadIdx.y] * b_shared[19][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[20][threadIdx.y] * b_shared[20][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[21][threadIdx.y] * b_shared[21][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[22][threadIdx.y] * b_shared[22][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[23][threadIdx.y] * b_shared[23][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[24][threadIdx.y] * b_shared[24][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[25][threadIdx.y] * b_shared[25][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[26][threadIdx.y] * b_shared[26][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[27][threadIdx.y] * b_shared[27][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[28][threadIdx.y] * b_shared[28][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[29][threadIdx.y] * b_shared[29][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[30][threadIdx.y] * b_shared[30][threadIdx.x];
			accum_shared[threadIdx.y][threadIdx.x] += aTT_shared[31][threadIdx.y] * b_shared[31][threadIdx.x];
			__syncthreads();
		}
		// multiply accum x a, add to c
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][0] * a_shared[0][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][1] * a_shared[1][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][2] * a_shared[2][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][3] * a_shared[3][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][4] * a_shared[4][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][5] * a_shared[5][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][6] * a_shared[6][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][7] * a_shared[7][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][8] * a_shared[8][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][9] * a_shared[9][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][10] * a_shared[10][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][11] * a_shared[11][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][12] * a_shared[12][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][13] * a_shared[13][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][14] * a_shared[14][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][15] * a_shared[15][threadIdx.x];
		
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][16] * a_shared[16][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][17] * a_shared[17][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][18] * a_shared[18][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][19] * a_shared[19][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][20] * a_shared[20][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][21] * a_shared[21][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][22] * a_shared[22][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][23] * a_shared[23][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][24] * a_shared[24][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][25] * a_shared[25][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][26] * a_shared[26][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][27] * a_shared[27][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][28] * a_shared[28][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][29] * a_shared[29][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][30] * a_shared[30][threadIdx.x];
		c_shared[threadIdx.y][threadIdx.x] += accum_shared[threadIdx.y][31] * a_shared[31][threadIdx.x];
		__syncthreads();
	}
	// save c to global
	Ce[c_idx] = c_shared[threadIdx.y][threadIdx.x];
}

//////////////////////////////////////////////////////////////////////////
////Task 3:  calculate the Frobenius norm of a matrix
////The definition of F-norm for a matrix is square root of (the sum of squares of all the matrix elements), i.e., F=sqrt(sum_(A_ij^2))
////See the definition: https://mathworld.wolfram.com/FrobeniusNorm.html
//////////////////////////////////////////////////////////////////////////

////Please write your own kernel function here, and call it in the function Test_F_Norm_On_GPU to test its correctness and performance
__global__ void F_Norm_On_GPU_Lazy(const float* Ae, float* sum)
{
	// lazy man's method for reference
	__shared__ float a_shared[16][16];
	int thr_per_block = blockDim.y * blockDim.x;
	int idx = blockIdx.y*gridDim.x*thr_per_block + threadIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
	
	float element = Ae[idx];
	a_shared[threadIdx.y][threadIdx.x] = element * element;
	atomicAdd(&sum[0], a_shared[threadIdx.y][threadIdx.x]);
}

__global__ void F_Norm_On_GPU(const float* Ae, float* Be, bool round1)
{
	extern __shared__ float data[];
	int idx = blockIdx.x*blockDim.x*2 + threadIdx.x;
	// use 2 registers
	float num1 = Ae[idx];
	float num2 = Ae[idx + blockDim.x]; // offset by stride is better for alignment

	// only square first time
	if (round1) {
		num1 *= num1;
		num2 *= num2;
	}
	// add two elements into one shared index
	data[threadIdx.x] = num1 + num2;
	__syncthreads();
	
	// from reduce4 in class notes
	for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
		if(threadIdx.x < s){
			data[threadIdx.x]+=data[threadIdx.x+s];
		}
		__syncthreads();
	}
	if (threadIdx.x == 0) Be[blockIdx.x] = data[0];
}
////Congratulations, your tasks are all finished!
//////////////////////////////////////////////////////////////////////////


////Here are the test functions for your three kernel implementations

ofstream out;

__host__ void Test_Matrix_Multiplication_AB_On_GPU(const Matrix& A,const Matrix& B,Matrix& C)
{
	//// Load A and B to device memory
	Matrix A_on_dev(A.m,A.n,false);
	A_on_dev=A;
	Matrix B_on_dev(B.m,B.n,false);
	B_on_dev=B;

	//// Allocate C in device memory
	Matrix C_on_dev(A_on_dev.m,B_on_dev.n,false);

	hipEvent_t start,end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	float gpu_time=0.0f;
	hipDeviceSynchronize();
	hipEventRecord(start);

	//// Invoke kernel
	const int block_size=32;
	const int block_num_x=C.m/block_size;
	const int block_num_y=C.n/block_size;

	#ifdef POORMAN
	Matrix_Multiplication_AB_Kernel_Poorman<<<dim3(block_num_x,block_num_y),dim3(block_size,block_size)>>>(A_on_dev.elements_on_dev,B_on_dev.elements_on_dev,C_on_dev.elements_on_dev,A_on_dev.m,A_on_dev.n,B_on_dev.n);
	#endif

	#ifndef POORMAN
	Matrix_Multiplication_AB_Kernel_Your_Version<<<dim3(block_num_y,block_num_x),dim3(block_size,block_size)>>>(A_on_dev.elements_on_dev,B_on_dev.elements_on_dev,C_on_dev.elements_on_dev,A_on_dev.m,A_on_dev.n,B_on_dev.n);
	#endif

	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&gpu_time,start,end);
	printf("\nGPU runtime for matrix multiplication AB: %.4f ms\n",gpu_time);
	hipEventDestroy(start);
	hipEventDestroy(end);

	//// Transfer data back to CPU
	C=C_on_dev;

	out<<"T1: "<<gpu_time<<endl;
}

__host__ void Test_Matrix_Multiplication_ATBA_On_GPU(const Matrix& A,const Matrix& B,Matrix& C)
{
	//// Load A and B to device memory
	Matrix A_on_dev(A.m,A.n,false);
	A_on_dev=A;
	Matrix B_on_dev(B.m,B.n,false);
	B_on_dev=B;
	
	//// Allocate C in device memory
	Matrix C_on_dev(A_on_dev.n,A_on_dev.n,false);

	hipEvent_t start,end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	float gpu_time=0.0f;
	hipDeviceSynchronize();
	hipEventRecord(start);

	//// Invoke kernel
	const int block_size=32;
	const int block_num_x=C.m/block_size;
	const int block_num_y=C.n/block_size;


	#ifdef POORMAN
		Matrix_Multiplication_ATBA_Kernel_Poorman<<<dim3(block_num_x,block_num_y),dim3(block_size,block_size)>>>(A_on_dev.elements_on_dev,B_on_dev.elements_on_dev,C_on_dev.elements_on_dev,A_on_dev.m,A_on_dev.n);
	#endif

	#ifndef POORMAN
	////NOTICE: You do not have to use the block_size I specified here. You may customize the size of your grid and blocks for better performance.
	Matrix_Multiplication_ATBA_Kernel_Your_Version<<<dim3(block_num_y,block_num_x),dim3(block_size,block_size)>>>(A_on_dev.elements_on_dev,B_on_dev.elements_on_dev,C_on_dev.elements_on_dev,A_on_dev.m,A_on_dev.n);
	#endif
	

	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&gpu_time,start,end);
	printf("\nGPU runtime for matrix multiplication ATBA: %.4f ms\n",gpu_time);
	hipEventDestroy(start);
	hipEventDestroy(end);

	//// Transfer data back to CPU
	C=C_on_dev;

	out<<"T2: "<<gpu_time<<endl;
}

__host__ void Test_Matrix_F_Norm_On_GPU(const Matrix& A, float& norm)
{
	//// Load A and B to device memory
	Matrix A_on_dev(A.m,A.n,false);
	A_on_dev=A;

	hipEvent_t start,end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	float gpu_time=0.0f;
	hipDeviceSynchronize();
	hipEventRecord(start);


	#ifdef POORMAN // atomic add

	//// Invoke kernel
	const int block_size=16;
	const int block_num_x=A.n/block_size;
	const int block_num_y=A.m/block_size;

	float *sum_dev = nullptr;
	cudaMalloc((void**)&sum_dev, sizeof(float));
	F_Norm_On_GPU_Lazy<<<dim3(block_num_x,block_num_y), dim3(block_size,block_size)>>>(A_on_dev.elements_on_dev, sum_dev);
	float *sum_host = (float *)malloc(4);
	cudaMemcpy(sum_host, sum_dev, sizeof(float), cudaMemcpyDeviceToHost);
	cudaFree(sum_dev);

	norm = sqrt(*sum_host);
	free(sum_host);

	#endif // ifdef

	#ifndef POORMAN // parallel reduction

	const int r1_blocks = A.m;
	const int r1_threads = A.n / 2;
	const int r2_threads = A.m / 2;
	float *B_dev = nullptr;
	hipMalloc((void**)&B_dev, A.m * sizeof(float));
	F_Norm_On_GPU<<<r1_blocks, r1_threads, r1_threads*sizeof(float)>>>(A_on_dev.elements_on_dev, B_dev, true);
	F_Norm_On_GPU<<<1, r2_threads, r2_threads*sizeof(float)>>>(B_dev, B_dev, false);

	float result = 0;
	hipMemcpy(&result,B_dev,sizeof(float),hipMemcpyDeviceToHost);
	norm = sqrt(result);
	hipFree(B_dev);
	#endif // ifndef
	
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&gpu_time,start,end);
	printf("\nGPU runtime for F norm: %.4f ms\n",gpu_time);
	hipEventDestroy(start);
	hipEventDestroy(end);

	out<<"T3: "<<gpu_time<<endl;
}

int main()
{
	if(name::team=="Team_X"){
		printf("\nPlease specify your team name and team member names in name::team and name::author to start.\n");
		return 0;
	}

	std::string file_name=name::team+"_competition_1_matrix.dat";
	out.open(file_name.c_str());

	if(out.fail()){
		printf("\ncannot open file %s to record results\n",file_name.c_str());
		return 0;
	}

	//////////////////////////////////////////////////////////////////////////
	////NOTICE: We may use a different set of parameters to evaluate your code.
	////So please test your functions with different size and initial values.
	//////////////////////////////////////////////////////////////////////////

	const int m=512;
	const int n=2048;
	const int p=1024;

	Matrix h_A(m,n);
	for(int i=0;i<m;i++){
		for(int j=0;j<n;j++){
			h_A(i,j) = 1;
		}
	}

	Matrix h_B(n,p);
	for(int i=0;i<n;i++){
		for(int j=0;j<p;j++){
			h_B(i,j) = 1;
		}
	}

	Matrix h_C(m,p);

	Matrix h_B2(m,m);
	for(int i=0;i<m;i++){
		for(int j=0;j<m;j++){
			h_B2(i,j) = 1;
		}
	}

	Matrix h_C2(n,n);

	Test_Matrix_Multiplication_AB_On_GPU(h_A,h_B,h_C);
	cout<<"AB result: "<<h_C(h_C.m/2,h_C.n/2)<<endl;
	out<<"R1: "<<h_C(h_C.m/2,h_C.n/2)<<endl;

	Test_Matrix_Multiplication_ATBA_On_GPU(h_A,h_B2,h_C2);
	cout<<"ATBA result: "<<h_C2(h_C2.m/3,h_C2.n/3)<<endl;
	out<<"R2: "<<h_C2(h_C2.m/3,h_C2.n/3)<<endl;

	float f_norm=0.f;
	Test_Matrix_F_Norm_On_GPU(h_A,f_norm);
	cout<<"F-norm result: "<<f_norm<<endl;
	out<<"R3: "<<f_norm<<endl;

	return 0;
}
